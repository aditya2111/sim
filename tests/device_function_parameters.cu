
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel2(int a) {
    printf("%d\n",a);
}

__global__ void kernel () {
    int a = 2;
    kernel2 <<< 1, 1 >>> (a);
}

int main(){
    kernel <<< 1, 1 >>> ();
    hipDeviceSynchronize();
    return 0;
}
