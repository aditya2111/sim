
#include <hip/hip_runtime.h>
#include <stdio.h>

__managed__ int store;

__global__
void kernel (int *a) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    a[0] += id;
    __syncthreads();
    printf("current thread : %d\ta[0]=%d\n",id,a[0]);
}

int main() {

    int s[1] = {0};
    int *d_s;

    hipMalloc(&d_s, sizeof(int) * 1);
    hipMemcpy(d_s, s, sizeof(int) * 1, hipMemcpyHostToDevice);

    kernel <<< 2, 3 >>> (d_s);
    hipDeviceSynchronize();

    hipMemcpy(s, d_s, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("%d\n", s[0]);
    return 0;
}
