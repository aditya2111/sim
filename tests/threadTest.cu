
#include <hip/hip_runtime.h>
#include <stdio.h>

__managed__ int store;

__global__
void kernel (int *a) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    a[0] += id;
    __syncthreads();
    printf("current thread : %d\ta[0]=%d\n",id,a[0]);
}

__global__
void blockReduction (int *a, int *ans) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    int tid = threadIdx.x;

    extern __shared__ int s[];
    s[tid] = a[id];
    __syncthreads();



}


// Thrust :: reduce.

int main() {

    int s[1] = {0};
    int *d_s;

    hipMalloc(&d_s, sizeof(int) * 1);
    hipMemcpy(d_s, s, sizeof(int) * 1, hipMemcpyHostToDevice);

    kernel <<< 2, 3 >>> (d_s);
    hipDeviceSynchronize();

    hipMemcpy(s, d_s, sizeof(int) * 1, hipMemcpyDeviceToHost);

    printf("%d\n", s[0]);
    return 0;
}
