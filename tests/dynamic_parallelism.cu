#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 5
#define nl printf("\n")

__global__ void kernel2 (int *a) {
    printf("child kernel!\n");
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[child]blockId : %d\n", blockIdx.x);
    a[id] = 24;
}

__global__ void kernel (int *a) {
    nl;
    printf("parent kernel!\n");
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    a[id] = id + 245;

    // current array 
    printf("current array : ");
    for (int i = 0; i < N; i++) {
        printf("%d ", a[i]);
    }nl;

    //dim3 gridDimension (N, N, 1); // x = N , y = N , z = 1

    // cuda dynamic parallelism
    kernel2 <<< N, 1 >>> (a); // 1024 * 23

    //printf("kernel2() executed!\n");   
}

void print (int *a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d ", a[i]);
    }
    nl;
}

int main() {
    int *a;
    int size = sizeof(int) * N;
    hipMallocManaged (&a, size);
    for (int i = 0; i < N; i++) {
        a[i] = i + 1;
    }

    printf("original : ");
    print(a, N);

    int deviceId;
    hipGetDevice(&deviceId);
    
    hipMemPrefetchAsync(a, size, deviceId);

    /* KERNEL CALL */
    kernel <<< 1, 1 >>> (a);
    hipDeviceSynchronize();
    nl;
    printf("converted array : ");   print(a, N);


    hipFree(a);
    return 0;
}

