
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 5


__global__ void kernel2 (int *a) {
    printf("[blockIdx.x]%d [gridDim.x]%d [threadIdx.x]%d\n",blockIdx.x, gridDim.x, threadIdx.x);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[kernel2()]id : %d\n", id);
    __syncthreads();

    a[id] = 24;
}

__global__ void kernel (int *a) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    printf("[kernel()]id : %d\n", id);
    a[id] = id + 1; 
    
    dim3 gridDimension (N, N, 1); // x = N , y = N , z = 1

    // cuda dynamic parallelism
    kernel2 <<<gridDimension, 1>>> (a); // 1024 * 23

    //printf("kernel2() executed!\n");   
}

int main() {
    int a[N];
    for(int i = 0; i < N; i++)
        a[i] = i + 1;

    int *device_a;
    hipMalloc(&device_a, sizeof(int) * N);
    hipMemcpy(device_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    kernel <<< 1, 1 >>> (device_a);
    //cudaDeviceSynchronize();
    hipMemcpy(a, device_a, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipFree(device_a);
    printf("\n");
    for(int i = 0; i < N; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
    return 0;
}

