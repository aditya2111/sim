#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10000
#define intu unsigned int

__global__ void kernel(intu* arr, intu* s) {
    __shared__ intu t;
   if(threadIdx.x == 0) t = 0;
    t += arr[threadIdx.x];
    __syncthreads();
    if(threadIdx.x == N-1)
        s[0] = t;
}

#define blockSize 1
__global__ void sumCommSingleBlock(const intu *a, intu *out) {
    intu idx = threadIdx.x;
    intu sum = 0;
    for (intu i = idx; i < N; i += blockSize)
        sum += a[i];
    __shared__ intu r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (intu size = blockSize/2; size>0; size/=2) { //uniform
        if (idx<size)
            r[idx] += r[idx+size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}
int main() {
    intu a[N];
    for(int i = 0; i < N; i++) a[i] = 1;
    intu* d_a;
    hipMalloc(&d_a, sizeof(intu)*N);
    hipMemcpy(d_a,a,sizeof(intu)*N, hipMemcpyHostToDevice);
    intu sum[1]; sum[0] = 0;
    intu *d_sum;
    hipMalloc(&d_sum, sizeof(intu));
    hipMemcpy(d_sum, sum, sizeof(intu), hipMemcpyHostToDevice);
    //kernel<<<1,N>>>(d_a, d_sum);
    sumCommSingleBlock(d_a, d_sum);
    hipMemcpy(sum, d_sum, sizeof(int), hipMemcpyDeviceToHost); 
    printf("summation : %d\n", sum[0]);
    return 0;
}
