#include "hip/hip_runtime.h"
#include <thrust/count.h>
__device__ int hh;
__global__ void print (int *a) {
    int id = threadIdx.x;
    a[id] = id + 1;
    hh += id + 1;

    __syncthreads();
}


__global__ void kernel () {
    int *a;
    //a = new int[5];
    a = (int*) malloc(sizeof(int) * 5);
    hh = 0;
    
    int size = 5;


    printf("array created !\n");
    print <<< 1, size >>> (a);
    for(int i = 0; i < size; i++) {
        printf("%d ", a[i]);
    }
    printf("\nh : %d\n", hh);
    __syncthreads();
}

int main(){ 
    int arr[] = {0, 1, 1, 1, 1, 0, 0, 0, 1, 0, 1, 0};
    int size = sizeof(arr) / sizeof(int);
    
    kernel <<< 1, 1 >>> ();
    hipDeviceSynchronize();
    return 0; 
}
