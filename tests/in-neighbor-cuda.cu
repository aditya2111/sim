#include "hip/hip_runtime.h"
#include <stdio.h>

void see(int *a, int size) {
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            printf("%d ", a[i * size + j]); 
        }
        printf("\n");
    }
}

__global__ void kernel (int *graph, int *in_neighbours, int *node, int* index, int *vertex) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("thread id : %d\n", id); 
    __syncthreads();
    in_neighbours[id] = (graph[id * vertex[0] + node[0]] == 1);
}

int main() {
    int edges, vertices;
    scanf("%d%d", &vertices, &edges);
    
    int graphSize = vertices * vertices;
    printf("edges : %d \t vertices : %d\n",edges, vertices);
    printf("graph size : %d\n", graphSize);
    int Graph[graphSize] = {0};
    
    for(int i = 0; i < edges; i++) {
        int a, b;
        scanf("%d%d", &a, &b);
        Graph[a * vertices + b] = 1;
    }

    printf("graph : \n");
    see(Graph, vertices);
    
    int *device_graph;
    hipMalloc(&device_graph, sizeof(int) * graphSize);
    hipMemcpy(device_graph, Graph, sizeof(int) * graphSize, hipMemcpyHostToDevice);
        
    int *device_vertices;
    int v[1] = {vertices};
    hipMalloc(&device_vertices, sizeof(int));
    hipMemcpy(device_vertices, v, sizeof(int), hipMemcpyHostToDevice); 

    int in_neighbours[vertices] = {0};
    int *device_in_neighbours;
    hipMalloc(&device_in_neighbours, sizeof(int) * vertices);
    hipMemcpy(device_in_neighbours, in_neighbours, sizeof(int) * vertices, hipMemcpyHostToDevice);

    int index[1] = {0};
    int *device_index;
    hipMalloc(&device_index, sizeof(int));
    hipMemcpy(device_index, index, sizeof(int), hipMemcpyHostToDevice);
        
    int node[1];
    printf("enter the node for which to calculate in-neighbours: ");
    scanf("%d",&node[0]);
    printf("node entered : %d\n", node[0]);
    int *device_node;
    hipMalloc(&device_node, sizeof(int));
    hipMemcpy(device_node, node, sizeof(int), hipMemcpyHostToDevice);
    
    int n_threads = vertices;
    printf("kernel call\n");
    kernel <<<1, n_threads>>> (device_graph, device_in_neighbours, device_node, device_index, device_vertices);
    hipMemcpy(in_neighbours, device_in_neighbours, sizeof(int) * vertices, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < sizeof(in_neighbours)/sizeof(int); i++) {
        printf("%d ",in_neighbours[i]); 
    }printf("\n"); 
    return 0;
}
