
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel2 () {
    printf("kernel2() called!\n");
}

__global__ void kernel1 () {
    printf("kernel1() called!\n");
    kernel2 <<< 1, 2 >>> ();
    printf("kernel2() returned!\n");
}

int main() {
    kernel1 <<< 1, 1 >>> ();
    hipDeviceSynchronize();
    return 0;    
}
