
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel () {
    if(threadIdx.x % 2 == 0)
        return;
    printf("thread : %d\n", threadIdx.x);
}

int main() {
    kernel <<< 1, 4 >>> ();
    hipDeviceSynchronize();
    return 0;
}
