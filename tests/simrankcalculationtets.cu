#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
using namespace std;
__managed__ int noOfVertices;
__managed__ double cal;

__global__
void inNeighboursParallel (int *graph, int *in_neighbours, int N, int node) {
	int id = threadIdx.x + (blockDim.x * blockIdx.x);
	for (int i = id; i < N; i += (gridDim.x * blockDim.x)) {
		in_neighbours[i] = graph[i * N + node];
	}
}


void printInNeighbours (int *inNeighbours, int size) {
	for (int i = 0; i < size; i++) {
		for (int j = 0; j <= size; j++) {
			printf ("%d ", inNeighbours[i * (size+1) + j]);
		}printf("\n");
	}
}

int *allInNeighbours (int *graph, int vertexCount, int *inNeighbours) {
	//printf("alliNNeighbours().\n");

	int deviceId;
	hipGetDevice (&deviceId);

	int noOfSMs, warpSize;
	hipDeviceGetAttribute (&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	hipDeviceGetAttribute (&warpSize, hipDeviceAttributeWarpSize, deviceId);

	//printf ("cuda parameters --> deviceId(%d), noOfSMs(%d).\n", deviceId, noOfSMs);

	for (int i = 0; i < vertexCount; i++) {
		int threadCnt = 1023, blockCnt = noOfSMs * 1024;
		int *tmp;
		hipMallocManaged (&tmp, sizeof(int) * vertexCount);

		int *deviceGraph;
		hipMallocManaged (&deviceGraph, sizeof(int) * vertexCount * vertexCount);
		hipMemcpy (deviceGraph, graph, sizeof(int) * vertexCount * vertexCount, hipMemcpyHostToDevice);

		//cout << "curr node : " << i << "\n";

		hipMemPrefetchAsync (tmp, sizeof(int) * vertexCount, deviceId);

		inNeighboursParallel <<< blockCnt, threadCnt >>> (deviceGraph, tmp, vertexCount, i);
		hipDeviceSynchronize();

		for (int j = 0; j < vertexCount; j++) {
			inNeighbours[i * (vertexCount + 1) + j] = tmp[j];
		}
		int one=0;
		for (int f = 0; f < vertexCount; f++)
			one += (inNeighbours[i * (vertexCount+1) + f] == 1);

		// store count.
		inNeighbours[i * (vertexCount+1) + vertexCount] = one;
		//printInNeighbours (inNeighbours, vertexCount);

	}
	return inNeighbours;
}


int* GraphInput () {
	int edges;
    //int noOfVertices;
    cin >> noOfVertices >> edges;
	int from, to, id=0, N=noOfVertices;
	printf("\ngraph config : \n\tno of vertices: %d\n\tno of edges : %d\n", noOfVertices, edges);
	int* graph;
	//graph = (int*)calloc(noOfVertices * noOfVertices, sizeof(int));
	hipMallocManaged(&graph, sizeof(int) * noOfVertices * noOfVertices);

	while (id < edges) {
        cin >> from >> to;
		graph[from * N + to] = 1;
		++id;
	}
	return graph;
}


// __global__
// void kernel (double *simrank, int from, int to, int *inNeighbours, int *graph) {
// 	int id = threadIdx.x + (blockIdx.x * blockDim.x);
// 	int stride = gridDim.x * blockDim.x;
// 	for (int i =  id; i <=
// }


int main() {
    freopen64("input.txt", "r", stdin);
    int *graph = GraphInput();

    int inNeighboursSize = sizeof(int) * (noOfVertices * (noOfVertices + 1));
	int *inNeighbours;
	hipMallocManaged (&inNeighbours, inNeighboursSize);
    inNeighbours = allInNeighbours (graph, noOfVertices, inNeighbours);
	printf("in-neighbours : \n");
    printInNeighbours (inNeighbours, noOfVertices);

    double *simrank;
    hipMallocManaged (&simrank, sizeof(double) * noOfVertices * noOfVertices);

    for (int i = 0; i < noOfVertices; i++) {
        for (int j = 0; j < noOfVertices; j++) {
            simrank[i * noOfVertices + j] = (1.0 * (i == j)) + 0.0;
        }
    }
    printf("before calculating simrank :\n");
	for (int i = 0; i < noOfVertices; i++) {
		for (int j = 0; j < noOfVertices; j++) {
			printf("%lf ", simrank[i * noOfVertices + j]);
		}printf("\n");
	}

    cal=0.0;
    /*kernel <<< noOfVertices * noOfVertices, 1024 >>> (simrank, 2, 3, inNeighbours, graph);
    hipDeviceSynchronize();
	*/
    return 0;
}
