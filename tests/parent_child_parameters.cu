#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10
#define newline printf("\n")

__global__ void kernel2 (int *device_a) {
   int id = threadIdx.x;
   device_a[id] = 10; 
}

__global__ void kernel (int *device_a) {
    int id = threadIdx.x;
    printf("original : ");
    for (int i = 0; i < N; i++) {
        printf("%d ", device_a[i]);
    }newline;

    printf("kernel2()\n");
    kernel2 <<< 1, N >>> (device_a); // child kernel
    hipDeviceSynchronize();

    printf("after updating: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", device_a[i]);
    }newline;

}

int main() {
    int a[N];
    for (int i = 0; i < N; i++) a[i] = -1; 
    int *device_a;
    hipMalloc(&device_a, sizeof(int) * N);
    hipMemcpy(device_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
     
    kernel <<< 1, 1 >>> (device_a);
    hipMemcpy (a, device_a, sizeof(int) * N, hipMemcpyDeviceToHost);

    return 0;
}
