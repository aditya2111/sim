#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <unordered_map>
#include "convergeGPU.h"
#include <fstream>
#include <stdio.h>
#include "array_operations.h"
#include <hip/hip_runtime_api.h>
#define d_ for(int i = 0; i < 100000; i++)
#define matrix_INT vector<vector<int>>
#define matrix_DOUBLE vector<vector<double>>
#define ROW_INT vector<int>
#define ROW_DOUBLE vector<double>

/* File Output
ofstream fout;
fout.open("output_simrank.txt");
****************/

void Message() {
    printf("Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : 0.9\n\t3. [No. of Iterations] : 1000\n");
}

/* GPU Kernel */
__global__ void calculateSimRankPair (int *A, int *B, double *simrank, double *summation, int* n_Vertices) {
    atomicAdd(&summation[0], simrank[A[blockIdx.x] * n_Vertices[0] + B[threadIdx.x]]);
}
/*************/


double totalKernelTime = 0.0;
void SimRankForAllNodes(int iteration, double* SimRank, int** Graph, int n_vertices, double confidence_value) {
   double* tmpSimrank = createArray_D(n_vertices); // default initialisation with 0.0 as array values. 
   
   /* Kernel Timing */
   clock_t startKernel, endKernel;
   //double totalKernelTime = 0.0;

   for(int i = 0; i < n_vertices; i++) {
       //hipStream_t streams[n_vertices];
       for(int j = 0; j < n_vertices; j++) {
           // GPU Computation.
           /* base condition: [1] Same Node */
           if(i == j) { 
               tmpSimrank[i * n_vertices + j] = 1.0;
               //printf("base case - 1[same node]\n");
               continue;
           }
           /* this code has some problem */
           int ia_size, ib_size;
           int* I_A = findInNeighbors(i, Graph, n_vertices, &ia_size);
           int* I_B = findInNeighbors(j, Graph, n_vertices, &ib_size);
           /******************************/
           /* Normalisation Factor */ 
           double normalisation_factor = confidence_value / (ia_size * ib_size);
            
           /* Base Condition : [2] No In_Neighbours of one of the edges */
           if(ia_size == 0 || ib_size == 0) {
               tmpSimrank[i * n_vertices + j] = 0.0;
               //printf("base case - 2[no in-neighbors]\n");
               continue;
           }

           /* GPU PARAMETERS */
           int n_CUDA_threads = ib_size;
           int n_CUDA_blocks = ia_size;

           //printf("GPU PARAMETERS :\t # of blocks : %d\t # of threads : %d\n", n_CUDA_blocks, n_CUDA_threads);
           /******************/ 

           /* not used.
              int nodes[2];
           nodes[0] = i;
           nodes[1] = j;

           int* device_nodes;
           hipMalloc(&device_nodes, sizeof(int) * 2); // 2 integers are stored, the #from node and the #to node
           hipMemcpy(device_nodes, nodes, sizeof(int) * 2, hipMemcpyHostToDevice);
           */
           int* d_I_A, *d_I_B;
           
           /* nv profiling */
           //hipProfilerStart();
           
           hipMalloc(&d_I_A, sizeof(int) * ia_size);
           hipMalloc(&d_I_B, sizeof(int) * ib_size);
           hipMemcpy(d_I_A, I_A, sizeof(int) * ia_size, hipMemcpyHostToDevice);
           hipMemcpy(d_I_B, I_B, sizeof(int) * ib_size, hipMemcpyHostToDevice);
            
           int n_simrank = n_vertices * n_vertices;
           double* d_simrank;
           hipMalloc(&d_simrank, sizeof(double) * n_simrank);
           hipMemcpy(d_simrank, SimRank, sizeof(double) * n_simrank, hipMemcpyHostToDevice);

           double tmp_ans[1]; tmp_ans[0] = 0.0;
           double* d_ans;
           hipMalloc(&d_ans, sizeof(double));        
           hipMemcpy(d_ans, tmp_ans, sizeof(double), hipMemcpyHostToDevice);

           int d_vert[1]; d_vert[0] = n_vertices;
           int *device_n_vertices;
           hipMalloc(&device_n_vertices, sizeof(int));
           hipMemcpy(device_n_vertices, d_vert, sizeof(int), hipMemcpyHostToDevice); 
 
           /* kernel call */
           startKernel = clock();
           calculateSimRankPair<<<n_CUDA_blocks, n_CUDA_threads>>>(d_I_A, d_I_B, d_simrank, d_ans, device_n_vertices);
           endKernel = clock();
           /***************/

           totalKernelTime += (double)(endKernel - startKernel) / CLOCKS_PER_SEC;
           //hipDeviceSynchronize(); 
           hipMemcpy(tmp_ans, d_ans, sizeof(double), hipMemcpyDeviceToHost);  

           // nv profiler end
           //hipProfilerStop();
           
           tmpSimrank[i * n_vertices + j] = tmp_ans[0] * normalisation_factor;
       }
   } 
   
    for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            SimRank[i * n_vertices + j] = tmpSimrank[i * n_vertices + j];
        }
    }

    // debug each iteration.
    //printf("Simrank updated!\n");
    /*for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            printf("%lf ", SimRank[i * n_vertices + j]);
        }
        printf("\n");
    }*/
}

void ComputeSimRankMatrix (int** Graph, int noOfVertices, int noOfEdges, int max_iterations, double confidence_value) {
    double* SimRank = (double*) malloc(sizeof(double) * noOfVertices * noOfVertices);
    int V = noOfVertices;
    // 1 st iterations.
    for(int i = 0; i < V; i++) {
        for(int j = 0; j < V; j++) {
            SimRank[i*V+j] = 0.0 + 1.0 * (i==j);
        }
    } 
    double normValue = 0.00;
    checkConvergence(SimRank, V, &normValue);
    // rest of the iterations/
    int k = 1;
    for(; k<max_iterations; k++) {
    //printf("iteration : #%d\n", k);
        /* below two functions are for plotting convergence graph */
        storeL2Norm(SimRank, noOfVertices);
        storel1Norm(SimRank, noOfVertices);

        SimRankForAllNodes(k, SimRank, Graph, noOfVertices, confidence_value);
        /* Checking Convergence of SimRank Matrix */ 
        if (k > 2 && checkConvergence(SimRank, noOfVertices, &normValue) == true) {
            break;
        }
    }
    //printf("Total Kernel Time : %.5f\n",totalKernelTime); 
    //printf("Converged on : %d\n",k);
    
    //printf("SimRank Algorithm Converged!\nFinal SimRank Matrix : \n");
    /*for(int i = 0; i < noOfVertices; i++) {
        for(int j = 0; j < noOfVertices; j++) {
            printf("%.4f ", SimRank[i*noOfVertices+j]);
        }printf("\n");
    }*/
    printf("\n");

}

int** TakeInput(int *V, int *E) {
    string filePath = "./tests/datasets/";
    string fileName = "watts_strogatz.txt";
    ifstream file(filePath + fileName);
    
    //ifstream file("input.txt");
    file >> *V;
    file >> *E;
    
    int n_vertices = *V, n_edges = *E;
    int from, to;
    int idx = 0;
    
    printf("\nEntered Graph Configuration : \n");
    printf("\tnoOfVertices: %d\n\tnoOfEdges: %d\n",*V,*E);  
    int** Graph = new int*[n_vertices+1];
    for(int i = 0; i < n_vertices; i++) {
        Graph[i] = new int[n_vertices+1];
    }
    while(idx < n_edges) {
        file >> from;
        file >> to;
        Graph[from][to] = 1;
        idx++; 
    }
    return Graph;
}

void TakeSimRankConfigurationInput(int &iterations, double &confidence) {
    printf("Enter no. of iterations[for default, input -1]: ");
    scanf("%d",&iterations);
    printf("Enter Confidence-Value[0-1, for default, input -1]: ");
    scanf("%lf",&confidence);

    if(iterations == -1) iterations = 1000;
    if(confidence == -1) confidence = 0.9;

    cout << "\n*SimRank Configuration Chosen: \n\tIterations: " << iterations << "\n\tConfidence Value: " << confidence << "\n";
}

int main() {
    Message();
    
    //Deleting the l1 norm parameters, for future creation.unavoidable 
    system("./delete_l1_l2.sh"); 
    
    // Graph Input.
    int noOfVertices, noOfEdges;
    
    int** Graph = TakeInput(&noOfVertices, &noOfEdges);

    // Input of SimRank Configuration parameters
    int noOfIterations;
    double confidence_value;
    TakeSimRankConfigurationInput(noOfIterations, confidence_value);
    
    // Time Calculation for Whole Computation.
    //clock_t startTime,endTime;
    
    //startTime = clock();
    ComputeSimRankMatrix(Graph, noOfVertices, noOfEdges, noOfIterations, confidence_value);
    //endTime = clock();
    
    //float time2 = (float)(endTime - startTime) / CLOCKS_PER_SEC;
    printf("[GPU]Time Elapsed in seconds: %.4f\n", totalKernelTime);
    
    //Generating Convergence Graph. 
    //system("python numpy_test.py");
    return 0;
}
