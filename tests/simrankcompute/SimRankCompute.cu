#include <hip/hip_runtime.h>
#include <device_atomic_functions.h>
#include <stdio.h>
#include <fstream>
using namespace std;
__managed__ int edgecnt, vertexcnt;
int* inputmatrix () {
	ifstream file("./graph.txt");
	file>>vertexcnt>>edgecnt;

	int from,to,i=0;
	int *graph;
	hipMallocManaged(&graph, sizeof(int) * vertexcnt * vertexcnt);
	
	while (i < vertexcnt) {
		file>>from;
		file>>to;

		graph[from*vertexcnt+to]=1;
		++i;
	}
	return graph;
}

__managed__ double ans;
__global__
void compute (int from, int to, double *simrank, int *graph) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);
	for (int i = id; i < vertexcnt; i++) {
		int f = i / vertexcnt;
		int t = i % vertexcnt;
		if (graph[from * vertexcnt + f] != 1 && graph[to * vertexcnt + t] != 1) return;
		printf("node : %d, %d\n", f, t);
		atomicAdd(&ans, simrank[f * vertexcnt + t]);
	}
}

int main() {
	int *graph;
	graph = inputmatrix();

	printf("graph : \n");
	for (int i = 0; i < vertexcnt; i++) {
		for (int j = 0; j < vertexcnt; j++) {
			printf("%d ", graph[i * vertexcnt + j]);
		}printf("\n");
	}
	
	double *simrank;
	hipMallocManaged(&simrank, sizeof(double) * vertexcnt * vertexcnt);

	for (int i = 0; i < vertexcnt; i++) {
		for (int j = 0; j < vertexcnt; j++) {
			simrank[i*vertexcnt+j] = (i == j) ? 1 : 0;	
		}
	}

	printf ("simrank matrix : \n");
	for (int i = 0; i < vertexcnt; i++) {
		for (int j = 0; j < vertexcnt; j++) {
			printf ("%lf ", simrank[i*vertexcnt+j]);
		}printf("\n");
	}
	
	int threads = 256;
	int blocks = vertexcnt * vertexcnt;

	ans=0;
	compute <<< blocks, threads >>> (2, 3, simrank, graph);
	hipDeviceSynchronize();
	
	printf ("ans : %lf\n", ans);

	return 0;
}
