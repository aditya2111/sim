
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define N 10000

__global__ void prefixSum (int *arr, int *st) {
    int id = threadIdx.x;
    if(id > 0) {
        arr[id] += arr[id-st[0]];
    }
}

__global__ void atomicSummation (int *arr, int* sum) {
    int id = threadIdx.x;
    atomicAdd(&sum[0], arr[id]);
}

int main() {
    int arr[N];
    for(int i = 0; i < N; i++) arr[i] = i + 1;
    int *d_arr;
    printf("original array : ");
    for(int i = 0; i < N; i++) printf("%d ", arr[i]);
    printf("\n");
    hipMalloc(&d_arr, sizeof(int)*N);
    hipMemcpy(d_arr, arr, sizeof(int)*N, hipMemcpyHostToDevice);
   
    clock_t start, end;
    double totalTime = 0.0;
    /* Method - 1 : prefix sum */
   /* printf("prefix sum: ");
    int limit = ceil(log(N) / log(2));
    //printf("log of %d: %d\n", N, limit);
    for(int i = 0; i < limit; i++) {
        int step = pow(2, i);
        int st[1]; st[0] = step;
        int *d_st;
        cudaMalloc(&d_st, sizeof(int));
        cudaMemcpy(d_st, st, sizeof(int), cudaMemcpyHostToDevice);
        //printf("step : %d\n", step);
        start = clock();
        prefixSum<<<1,N>>>(d_arr, d_st);
        end = clock();
        totalTime += (double) (end - start) / CLOCKS_PER_SEC;
        cudaDeviceSynchronize(); 
    }
    cudaMemcpy(arr, d_arr, sizeof(int)*N, cudaMemcpyDeviceToHost);
   // printf("prefix sum:\n");
    for(int i = 0; i < N; i++) printf("%d ", arr[i]);
    printf("\nSummation : %d\n",arr[N-1]);

    printf("Total Time : %lf\n",totalTime);
   */ 
    totalTime = 0.0;
    printf("\natomic operations\n");
    int sum[1]; sum[0] = 0;
    int* d_sum;
    hipMalloc(&d_sum, sizeof(int));
    hipMemcpy(d_sum, sum, sizeof(int), hipMemcpyHostToDevice);
    start = clock();
    atomicSummation<<<1, N>>>(d_arr, d_sum);
    end = clock();
    totalTime += (double)(end - start) / CLOCKS_PER_SEC;
    hipMemcpy(arr, d_arr, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
   // printf("final array :");
    //for(int i = 0; i < N; i++) printf("%d ", arr[i]);
    printf("\nTotal Time %lf\n", totalTime);

    printf("\nFinal Summation : %d\n", sum[0]);
   return 0;
}
