#include "hip/hip_runtime.h"
#include <stdio.h>



__global__ void sum (int *arr, int *sum, int *size){
    int id = threadIdx.x;
    int result = 0;
    kernel2 <<< 1, size[0] >>> (arr, result);
    __syncthreads();
    sum[0] = result;
}

int main() {
    int arr[] = {0, 1, 1, 0};
    int size = sizeof(arr) / sizeof(int);

    int *device_arr;
    hipMalloc(&device_arr, sizeof(int) * size);
    hipMemcpy(device_arr, arr, sizeof(int) * size, hipMemcpyHostToDevice);
    
    int *device_size;
    int s[1] = {size};
    hipMalloc(&device_size, sizeof(int)); 
    hipMemcpy(device_size, s, sizeof(int), hipMemcpyHostToDevice);

    int v[1] = {0};
    int *device_v;
    hipMalloc(&device_v, sizeof(int));
    hipMemcpy(device_v, v, sizeof(int), hipMemcpyHostToDevice);

    sum <<< 1, 1 >>> (device_arr, device_v, device_size);
    hipMemcpy(v, device_v, sizeof(int), hipMemcpyDeviceToHost);
        
    printf("sum : %d \n", v[0]);

    return 0;
}
