#include "hip/hip_runtime.h"
#include "./include/include_files.hpp"

__managed__ int noOfVertices;

void PreSetup () {
	system("truncate -s 0 ./data/l1Norm.txt"); // clearing up previous norms.
}

void ShowAlgoDefaults () {
    printf ("Default Configuration : \n\t1. [directed-graph]\n\t2. [confidence value] : %lf\n\t3. [max. no of iterations] : %d\n", defaultConfidenceValue, defaultMaxIterations);
}

int* GraphInput () {
	int edges;
	ifstream fileptr (DATASET_FOLDER + "graph_input.txt");
	fileptr >> noOfVertices >> edges;
	
	int from, to, id=0, N=noOfVertices;
	printf("\ngraph config : \n\tno of vertices: %d\n\tno of edges : %d\n", noOfVertices, edges);
	int* graph;
	//graph = createArray<int> (noOfVertices);
	graph = (int*)calloc(noOfVertices * noOfVertices, sizeof(int));
	
	while (id < edges) {
		fileptr >> from;
		fileptr >> to;
		graph[from * N + to] = 1;
		++id;
	}	
	return graph;
}


void CalculateSimrankUtil (double *SimrankCurrent, int *graph, int currentIteration, double ConfidenceValue) {
	int sizeOfSimrank = noOfVertices * noOfVertices;
	double *nextSimrank = createArray <double> (noOfVertices);
	double *device_nextSimrank;
	hipMalloc(&device_nextSimrank,sizeof(double) * sizeOfSimrank);
	hipMemcpy (device_nextSimrank, nextSimrank, sizeof(double) * sizeOfSimrank, hipMemcpyHostToDevice);

	int BlockCount, ThreadCount;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute (&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// PreCompute The In-Neighbours;
	int inNeighboursSize = sizeof(int) * (noOfVertices * (noOfVertices + 1));
	int *inNeighbours;
	hipMallocManaged (&inNeighbours, inNeighboursSize);

	inNeighbours = allInNeighbours (graph, noOfVertices, inNeighbours);

	//debugInNeighbours (inNeighbours, noOfVertices);

	int *pursuePairs;
	int noOfPairs = noOfVertices * noOfVertices;

	pursuePairs = calloc (noOfPairs * 2, sizeof(int)); // init to 0.

	BlockCount = noOfVertices * noOfVertices;
	ThreadCount = 1024;

	pursuePairs = storePairs (pursuePairs, noOfVertices, noOfPairs); // generate pairs.

	// update BlockCount and ThreadCount.
	BlockCount = noOfPairs * 2;
	ThreadCount = 1024;

	// allocating space in GPU
	int *device_Pairs;
	hipMalloc (&device_Pairs, sizeof(int) * noOfPairs * 2);
	hipMemcpy (device_Pairs, pursePPairs, sizeof(int) * noOfPairs * 2, hipMemcpyHostToDevice);

	int *device_graph;
	hipMalloc(&device_graph, sizeof(int) * noOfVertices * noOfVertices);
	hipMemcpy(device_graph, graph, sizeof(int) * noOfVertices, noOfVertices, hipMemcpyHostToDevice);



	kernel <<< BlockCount, ThreadCount >>> (device_Pairs, inNeighbours, device_graph, device_nextSimrank, );
	hipDeviceSynchronize();

	return;
}

void ComputeSimrank (int *graph, int MaxNoOfIterations, double ConfidenceValue) {
	double *simrank = createArray <double> (noOfVertices);
	for (int i = 0; i < noOfVertices; i++) {
		simrank[i * noOfVertices + i] = 1.0;
	}
	//seeMatrix <double> (simrank, noOfVertices); //--> simrank matrix at iteration-0.
	double scoreOfSimrankMatrix=0.0;
	// calculate score for iteration-0
	converge (simrank, noOfVertices, &scoreOfSimrankMatrix);
	printf ("score of simrank matrix at iteration 0 : %lf\n", scoreOfSimrankMatrix);
	int iteration=1, ConvergedPoint=INT_MAX;

	for ( ; iteration <= MaxNoOfIterations ; iteration++) {
		storeSimrankScore (simrank, noOfVertices);
		// calculating simrank.
		CalculateSimrankUtil (simrank, graph, iteration, ConfidenceValue);

		bool convergeflag = converge (simrank, noOfVertices, &scoreOfSimrankMatrix);
		if (iteration >= 3 && convergeflag == true) {
			ConvergedPoint = iteration;
			break;
		}
	}

	cout << "converged at : " << ConvergedPoint << "\n";
}



int main() {
	PreSetup();
	ShowAlgoDefaults();

	int *graph = GraphInput();
	
	seeMatrix<int> (graph, noOfVertices);

	int MaxIterations;
	double ConfidenceValue;
	simrankConfigInput (MaxIterations, ConfidenceValue);
	
	// compute simrank
	ComputeSimrank (graph, MaxIterations, ConfidenceValue);
	return 0;
}
