#include "hip/hip_runtime.h"
#include "./include/include_files.hpp"

__managed__ int noOfVertices;

void PreSetup () {
	system("truncate -s 0 ./data/l1Norm.txt"); // clearing up previous norms.
	system("truncate -s 0 ./gpu_output.txt"); // clearing previous simrank output.
}

void ShowAlgoDefaults () {
    printf ("Default Configuration : \n\t1. [directed-graph]\n\t2. [confidence value] : %lf\n\t3. [max. no of iterations] : %d\n", defaultConfidenceValue, defaultMaxIterations);
}

int* GraphInput () {
	int edges;
	ifstream fileptr (DATASET_FOLDER + "graph_input.txt");
	fileptr >> noOfVertices >> edges;
	
	int from, to, id=0, N=noOfVertices;
	printf("\ngraph config : \n\tno of vertices: %d\n\tno of edges : %d\n", noOfVertices, edges);
	int* graph;
	//graph = createArray<int> (noOfVertices);
	graph = (int*)calloc(noOfVertices * noOfVertices, sizeof(int));
	
	while (id < edges) {
		fileptr >> from;
		fileptr >> to;
		graph[from * N + to] = 1;
		++id;
	}	
	return graph;
}

__global__
void kernel (int *pursuePairs, int *inNeighbours, int *graph, double *simrank, double *prevSimrank, int count, double ConfidenceValue) {
	int id = threadIdx.x + (blockIdx.x * blockDim.x);
	int gridStride = blockDim.x * gridDim.x;
	//printf("kernel call -> id : %d\n", id);
	for (int i = id; i <= count; i += gridStride) {
		int from = pursuePairs[i * 2 + 0];
		int to = pursuePairs[i * 2 + 1];
		
		//printf("curr nodes : (%d, %d)\n",from,to);
			
		__syncthreads();
		double ans = simrank_utility(from, to, graph, inNeighbours, prevSimrank, noOfVertices, ConfidenceValue);
// 		printf("simrank ans : %lf\n", ans);
		simrank[from * noOfVertices + to] = ans;
	}
}


void CalculateSimrankUtil (double *SimrankCurrent, int *graph, int currentIteration, double ConfidenceValue) {
	int sizeOfSimrank = noOfVertices * noOfVertices;
	double *nextSimrank = createArray <double> (noOfVertices);
	double *device_nextSimrank;
	hipMalloc(&device_nextSimrank,sizeof(double) * sizeOfSimrank);
	hipMemcpy (device_nextSimrank, nextSimrank, sizeof(double) * sizeOfSimrank, hipMemcpyHostToDevice);

	double *device_currSimrank;
	hipMalloc(&device_currSimrank, sizeof(double) * sizeOfSimrank);
	hipMemcpy (device_currSimrank, SimrankCurrent, sizeof(double) * sizeOfSimrank, hipMemcpyHostToDevice);

	int BlockCount, ThreadCount;
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute (&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// PreCompute The In-Neighbours;
	int inNeighboursSize = sizeof(int) * (noOfVertices * (noOfVertices + 1));
	int *inNeighbours;
	hipMallocManaged (&inNeighbours, inNeighboursSize);
	//hipMemPrefetchAsync (inNeighbours, sizeof(int) * (noOfVertices * (noOfVertices + 1)), deviceId);
	inNeighbours = allInNeighbours (graph, noOfVertices, inNeighbours);

	//debugInNeighbours (inNeighbours, noOfVertices);

	int *pursuePairs;
	int noOfPairs = noOfVertices * noOfVertices;

	pursuePairs = (int*)calloc (noOfPairs * 2, sizeof(int)); // init to 0.

	BlockCount = noOfVertices * noOfVertices;
	ThreadCount = 1024;

	pursuePairs = storePairs (pursuePairs, noOfVertices, noOfPairs); // generate pairs.

	// update BlockCount and ThreadCount.
	BlockCount = noOfPairs;
	ThreadCount = 1024;

	// allocating space in GPU
	int *device_Pairs;
	hipMalloc (&device_Pairs, sizeof(int) * noOfPairs * 2);
	hipMemcpy (device_Pairs, pursuePairs, sizeof(int) * noOfPairs * 2, hipMemcpyHostToDevice);

	int *device_graph;
	hipMalloc(&device_graph, sizeof(int) * noOfVertices * noOfVertices);
	hipMemcpy(device_graph, graph, sizeof(int) * noOfVertices * noOfVertices, hipMemcpyHostToDevice);

	kernel <<< BlockCount, ThreadCount >>> (device_Pairs, inNeighbours, device_graph, device_nextSimrank, device_currSimrank, noOfPairs, ConfidenceValue);
	hipDeviceSynchronize();

	hipMemcpy (nextSimrank, device_nextSimrank, sizeof(double) * noOfVertices * noOfVertices, hipMemcpyDeviceToHost);

	// copy the new simrank.
	for (int i = 0; i < noOfVertices; i++) {
		for (int j = 0; j < noOfVertices; j++) {
			SimrankCurrent[i * noOfVertices + j] = nextSimrank[i * noOfVertices + j];
		}
	}
	
	// Verbose output
	/*printf("\n\n-----------");
	printf("current iteration %d simrank : \n", currentIteration);
	seeMatrix <double> (SimrankCurrent, noOfVertices);	
	*/
	return;
}

void ComputeSimrank (int *graph, int MaxNoOfIterations, double ConfidenceValue) {
	double *simrank = createArray <double> (noOfVertices);
	for (int i = 0; i < noOfVertices; i++) {
		simrank[i * noOfVertices + i] = 1.0;
	}
	//seeMatrix <double> (simrank, noOfVertices); //--> simrank matrix at iteration-0.
	double scoreOfSimrankMatrix=0.0;
	// calculate score for iteration-0
	converge (simrank, noOfVertices, &scoreOfSimrankMatrix);
	//printf ("score of simrank matrix at iteration 0 : %lf\n", scoreOfSimrankMatrix);
	int iteration=1, ConvergedPoint=INT_MAX;

	for ( ; iteration <= MaxNoOfIterations ; iteration++) {
		//printf ("score of simrank matrix at iteration %d : %lf\n", iteration, scoreOfSimrankMatrix);
		storeSimrankScore (simrank, noOfVertices);
		// calculating simrank.
		CalculateSimrankUtil (simrank, graph, iteration, ConfidenceValue);

		bool convergeflag = converge (simrank, noOfVertices, &scoreOfSimrankMatrix);
		if (iteration >= 3 && convergeflag == true) {
			ConvergedPoint = iteration;
			break;
		}
	}

	cout << "converged at : " << iteration << "\n";
	cout << "\nsimrank output stored @./gpu_output.txt\n"; 	
	ofstream store;
	store.open("gpu_output.txt", ios::app);
	for (int i = 0; i < noOfVertices; i++) {
		for (int j = 0; j < noOfVertices; j++) {
			store << simrank[i * noOfVertices + j] << " ";
			// printf("%lf ", simrank[i * noOfVertices + j]);
		}
		// printf("\n");
		store << "\n";
	}
	store.close();
}



int main() {
	PreSetup();
	ShowAlgoDefaults();

	int *graph = GraphInput();
	
	printf("adjacency matrix of graph :\n");
	seeMatrix<int> (graph, noOfVertices);

	int MaxIterations;
	double ConfidenceValue;
	simrankConfigInput (MaxIterations, ConfidenceValue);
	
	// compute simrank
	ComputeSimrank (graph, MaxIterations, ConfidenceValue);

	return 0;
}
