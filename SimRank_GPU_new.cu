#include "hip/hip_runtime.h"
#include "./include/include_files.hpp"

void defaultAlgoPara () {
    printf ("Default Configuration : \n\t1. [directed-graph]\n\t2. [confidence value] : %lf\n\t3. [max. no of iterations] : %d\n", defaultConfidenceValue, defaultMaxIterations);
}

int* GraphInput (int *vertices, int *edges) {
	ifstream fileptr (DATASET_FOLDER + "graph_input.txt");
	fileptr >> *vertices >> *edges;
	
	int from, to, id=0, N=*vertices;
	printf("\ngraph config : \n\tno of vertices: %d\n\tno of edges : %d\n", *vertices, *edges);
	int* graph;
	graph = createArray<int> (*vertices);
	
	while (id < *edges) {
		fileptr >> from;
		fileptr >> to;
		
		graph[from * N + to] = 1;
		++id;
	}	
	return graph;
}


int main() {
	defaultAlgoPara();
	int vertices, edges;
	int *graph = GraphInput(&vertices, &edges);
	
	int MaxIterations;
	double ConfidenceValue;
	simrankConfigInput (MaxIterations, ConfidenceValue);
	
	
	
	// compute simrank
	
	return 0;
}
