#include "hip/hip_runtime.h"
#include "convergeGPU.h"
#include "include_files.h"

extern void in_neighbour_calculation_cuda(int *graph, int *in_neighbour, int node, int n_vertices);

void ShowMessage() {
    cout << "Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : " << defaultConfidenceValue 
            << "\n\t3. [No. of Iterations] : " << defaultMaxIterations << "\n";
}

void calculateSimRankForEachPair () {
    return; 
}

void compute_simrank (int *graph, int noOfVertices, int noOfIterations, double confidenceValue) {
    double *simrank;
    initGraph<double> (simrank, noOfVertices, 0.0);

    double normValue=0.0; // for convergence calculation // donot change.
    int currentIteration = 1;


    while (currentIteration <= noOfIterations) {
        storeNorm(simrank, noOfVertices, "L1");
        storeNorm(simrank, noOfVertices, "L2");

        // after experimentation; min value of currerntIteration = 3 can be used.
        if (currentIteration > 3 && checkConvergence(simrank, noOfVertices, &normValue, "L1")) {
            // using L1 Norm for convergence
            break;
        }

        calculateSimRankForEachPair(); // for each (i, j) from |V x V|
        


        ++currentIteration;
    }


}


int *TakeGraphInput(int *vertices, int *edges, string fileName) {
    ifstream filePtr(DATASET_FOLDER + fileName);
    filePtr >> *vertices;
    filePtr >> *edges;

    int from, to, idx=0, N = *vertices;

    cout << "\nEntered Graph Configuration : \n\tnoOfVertices : " << *vertices << "\n\tnoOfEdges : " << *edges << "\n";

    // storing graph in unified memory.
    int *graph;
    hipMallocManaged(&graph, N * N);
    initGraph<int>(graph, N, 0); // initialise graph to zero

    while (idx < *edges) {
        filePtr >> from;
        filePtr >> to;
        graph[from * N + to] = 1;
        ++idx;
    }
    return graph;
}

int main() {
    ShowMessage();
    // system("./delete_l1_l2.sh");

    int noOfVertices, noOfEdges;
    int *Graph;
    Graph = TakeGraphInput(&noOfVertices, &noOfEdges, "graph_input.txt");
    seeGraph<int>(Graph, noOfVertices);

    int MaxNoOfIterations;
    double confidenceValue;
    TakeSimRankConfigurationInput(MaxNoOfIterations, confidenceValue);

    // compute_simrank (Graph, noOfVertices, MaxNoOfIterations, confidenceValue);

    // system("python numpy_test.py"); // generates convergence graph

    return 0;
}