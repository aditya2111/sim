#include "hip/hip_runtime.h"
#include "convergeGPU.h"
#include "include_files.h"
#include "cuda_operations_simrank.cuh"
#include <__clang_cuda_runtime_wrapper.h>
#include <system_error>

void ShowMessage() {
    cout << "Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : " << defaultConfidenceValue 
            << "\n\t3. [No. of Iterations] : " << defaultMaxIterations << "\n";
}

__managed__ double ConfidenceValue_;
__managed__ int ThreadCount_, BlockCount_;


__global__
void computeForAPairNodes (int *graph, int verticesCount, double *currentSimRankMtx, double *futureSimRankMtx, int *in_neighbours) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int sqVertices = verticesCount * verticesCount;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = id; i < verticesCount * verticesCount; i += gridStride) { 
        int from, to; // store nodes;
        from = i % verticesCount;
        to = i / verticesCount;

        futureSimRankMtx[from * verticesCount + from] = 1.0;
        futureSimRankMtx[to * verticesCount + to] = 1.0;
        
        // other kernel.
    

        int count_inNeighbours_FROM = calculateCountOfInNeighbours (in_neighbours, from, verticesCount);
        int count_inNeighbours_TO =  calculateCountOfInNeighbours (in_neighbours, to, verticesCount);
        
        int totalCount = count_inNeighbours_FROM * count_inNeighbours_TO;
        int N_THREADS, N_BLOCKS;
        
        N_THREADS = 1024;
        N_BLOCKS =  ceil (totalCount / N_THREADS);

        // CPU is faster. [experimental]
        computeFromInNeighbours (futureSimRankMtx, currentSimRankMtx, graph, verticesCount, in_neighbours, from, to);
    }
}

void calculateSimRankForEachPair (double *simrank, int n_vertices, int *graph, double confidenceValue) {
    // for each pair of nodes
    double *tmpSimRank;
    hipMallocManaged(&tmpSimRank, sizeof(double) * n_vertices * n_vertices);
    
    int total_threads = n_vertices * n_vertices;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, deviceId);
    int sqVertices = n_vertices * n_vertices;
    ThreadCount_ = 1024;
    BlockCount_ = ceil (sqVertices / ThreadCount_);


    int *InNeighbours = calculateAllInNeighbours(graph,n_vertices); // calculates and stores all the in-neighbours 


    computeForAPairNodes <<< BlockCount_, ThreadCount_ >>> (graph, n_vertices, simrank, tmpSimRank, InNeighbours);
    return; 
}

void compute_simrank (int *graph, int noOfVertices, int noOfIterations, double confidenceValue) {
    double *simrank;
    hipMallocManaged(&simrank, sizeof(double) * noOfVertices * noOfVertices);
    initGraph<double> (simrank, noOfVertices, 0.0);

    double normValue=0.0; // for convergence calculation // donot change.
    int currentIteration = 1;

    ConfidenceValue_ = confidenceValue;

    while (currentIteration <= noOfIterations) {
        storeNorm(simrank, noOfVertices, "L1");
        storeNorm(simrank, noOfVertices, "L2");

        // after experimentation; min value of currerntIteration = 3 can be used.
        if (currentIteration > 3 && checkConvergence(simrank, noOfVertices, &normValue, "L1")) {
            // using L1 Norm for convergence
            break;
        }

        calculateSimRankForEachPair(simrank, noOfVertices, graph, confidenceValue); // for each (i, j) from |V x V|
        ++currentIteration;
    }
}


int *TakeGraphInput(int *vertices, int *edges, string fileName) {
    ifstream filePtr(DATASET_FOLDER + fileName);
    filePtr >> *vertices;
    filePtr >> *edges;

    int from, to, idx=0, N = *vertices;

    cout << "\nEntered Graph Configuration : \n\tnoOfVertices : " << *vertices << "\n\tnoOfEdges : " << *edges << "\n";

    // storing graph in unified memory.
    int *graph;
    hipMallocManaged(&graph, N * N);
    initGraph<int>(graph, N, 0); // initialise graph to zero

    while (idx < *edges) {
        filePtr >> from;
        filePtr >> to;
        graph[from * N + to] = 1;
        ++idx;
    }
    return graph;
}

int main() {
    ShowMessage();
    // system("./delete_l1_l2.sh");

    int noOfVertices, noOfEdges;
    int *Graph;
    Graph = TakeGraphInput(&noOfVertices, &noOfEdges, "graph_input.txt");
    seeGraph<int>(Graph, noOfVertices);

    int MaxNoOfIterations;
    double confidenceValue;
    TakeSimRankConfigurationInput(MaxNoOfIterations, confidenceValue);

    // compute_simrank (Graph, noOfVertices, MaxNoOfIterations, confidenceValue);

    // system("python numpy_test.py"); // generates convergence graph

    return 0;
}