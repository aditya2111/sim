#include "hip/hip_runtime.h"
#include "array_operations.h"
#include "convergeGPU.h"
#include "include_files.h"
#include "cuda_operations_simrank.cuh"
// #include <__clang_cuda_runtime_wrapper.h>
#include <system_error>

void ShowMessage() {
    cout << "Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : " << defaultConfidenceValue 
            << "\n\t3. [No. of Iterations] : " << defaultMaxIterations << "\n";
}


__managed__ double ConfidenceValue_;
__managed__ int ThreadCount_, BlockCount_;


__global__
void computeForAPairNodes (int *graph, int verticesCount, double *currentSimRankMtx, double *futureSimRankMtx, int *in_neighbours) {
    // printf("blockIdx : %d\tThreadIdx : %d\n", threadIdx.x, blockIdx.x);
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // int sqVertices = verticesCount * verticesCount;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = id; i < verticesCount * verticesCount; i += gridStride) { 
        // cout << "i : " << i << "\n";
        // printf ("i : %d\n", i);
        int from, to; // store nodes;
        from = i % verticesCount;
        to = i / verticesCount;
        // printf ("current node pair [%d, %d]\n", from, to);

        futureSimRankMtx[from * verticesCount + from] = 1.0;
        futureSimRankMtx[to * verticesCount + to] = 1.0;
        
        int count_inNeighbours_FROM = calculateCountOfInNeighbours (in_neighbours, from, verticesCount);
        int count_inNeighbours_TO =  calculateCountOfInNeighbours (in_neighbours, to, verticesCount);
        
        // DEBUG-In_Neighbour count.
        // if (from == 10 && to == 12) {
        //     printf ("inNeighbours count; for %d : %d and %d : %d\n", from, count_inNeighbours_FROM, to, count_inNeighbours_TO);
        // }
        if (from == to) return;
        if (count_inNeighbours_FROM == 0 || count_inNeighbours_TO == 0) {
            // printf ("pair of nodes with zero In_neighbours : %d and %d\n", from, to);
            // printf ("thread blocked : %d\n", threadIdx.x);
            futureSimRankMtx[from * verticesCount + to] = 0.0;
            return;
        }

        double NORMALISATION_FACTOR = ConfidenceValue_ / (double)(count_inNeighbours_FROM * count_inNeighbours_TO);
        // if (from == 4 && to == 2)
        //     printf ("normalisation factor : %lf\n", NORMALISATION_FACTOR);

        // CPU [experimental] -> [faster than GPU]
        double simrank_computed_in_neighbour = computeFromInNeighbours (futureSimRankMtx, currentSimRankMtx, 
                                                                        graph, verticesCount, in_neighbours, 
                                                                        from, to, count_inNeighbours_FROM, count_inNeighbours_TO,
                                                                        NORMALISATION_FACTOR);
        // hipDeviceSynchronize();
        futureSimRankMtx[from * verticesCount + to] = NORMALISATION_FACTOR * simrank_computed_in_neighbour;
    }
}

void calculateSimRankForEachPair (double *simrank, int n_vertices, int *graph, double confidenceValue, int iterationCount) {
    // for each pair of nodes
    double *tmpSimRank;
    hipMallocManaged(&tmpSimRank, sizeof(double) * n_vertices * n_vertices);
    
    // int total_threads = n_vertices * n_vertices;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, deviceId);
    
    // THREAD CONFIG.
    int sqVertices = n_vertices * n_vertices;
    if (n_vertices > 1024)
        ThreadCount_ = 1024;
    else ThreadCount_ = n_vertices;
    BlockCount_ = ceil (sqVertices / ThreadCount_);
    if (BlockCount_ == 0) BlockCount_ = 1;

    int *InNeighbours;
    hipMallocManaged (&InNeighbours, sizeof(int) * sqVertices);
    calculateAllInNeighbours(graph,n_vertices,InNeighbours); // calculates and stores all the in-neighbours 

    // DEBUG.
    // seeGraph<int>(InNeighbours,n_vertices); // wrong.
    // printf ("in-neighbours for node - 12\n");
    // for (int i = 0; i < n_vertices; i++) {
    //     printf ("%d ", InNeighbours[12 * n_vertices + i]);
    // }printf ("\n");

    // printf ("ThreadCount : %d & BlockCount : %d\n", ThreadCount_, BlockCount_);
    // printf ("computeForAPairNodes() call.\n");

    computeForAPairNodes <<< BlockCount_, ThreadCount_ >>> (graph, n_vertices, simrank, tmpSimRank, InNeighbours);
    hipDeviceSynchronize();


    printf ("iteration : #%d \n", iterationCount);
    seeSimrank(tmpSimRank, n_vertices);
    copyArr<double>(tmpSimRank, simrank, n_vertices); // iteration complete. 

    return; 
}

void compute_simrank (int *graph, int noOfVertices, int noOfIterations, double confidenceValue) {
    double *simrank;
    hipMallocManaged(&simrank, sizeof(double) * noOfVertices * noOfVertices);
    initGraph<double> (simrank, noOfVertices, 0.0);

    double normValue=0.0; // for convergence calculation // donot change.
    int currentIteration = 1;

    ConfidenceValue_ = confidenceValue;

    while (currentIteration <= noOfIterations) {
        storeNorm(simrank, noOfVertices, "L1");
        storeNorm(simrank, noOfVertices, "L2");

        // after experimentation; min value of currerntIteration = 3 can be used.
        if (currentIteration > 3 && checkConvergence(simrank, noOfVertices, &normValue, "L1")) {
            // using L1 Norm for convergence
            break;
        }
        // printf ("calculateSimRankForEachPair() call.\n");
        calculateSimRankForEachPair(simrank, noOfVertices, graph, confidenceValue, currentIteration); // for each (i, j) from |V x V|
        ++currentIteration;
    }


    // un-comment for verbose output.
    // seeSimrank(simrank, noOfVertices); // print ans.

    // cout << "converged!\n";
}


int *TakeGraphInput(int *vertices, int *edges, string fileName) {
    ifstream filePtr(DATASET_FOLDER + fileName);
    filePtr >> *vertices;
    filePtr >> *edges;

    int from, to, idx=0, N = *vertices;

    cout << "\nEntered Graph Configuration : \n\tnoOfVertices : " << *vertices << "\n\tnoOfEdges : " << *edges << "\n";

    // storing graph in unified memory.
    int *graph;
    hipMallocManaged(&graph, N * N);
    initGraph<int>(graph, N, 0); // initialise graph to zero
 
    while (idx < *edges) {
        filePtr >> from;
        filePtr >> to;
        graph[from * N + to] = 1;
        ++idx;
    }
    return graph;
}

int main() {
    ShowMessage();
    // system("./delete_l1_l2.sh");

    int noOfVertices, noOfEdges;
    int *Graph;
    Graph = TakeGraphInput(&noOfVertices, &noOfEdges, "graph_input.txt");
    // seeGraph<int>(Graph, noOfVertices);

    int MaxNoOfIterations;
    double confidenceValue;
    TakeSimRankConfigurationInput(MaxNoOfIterations, confidenceValue);
    assert (MaxNoOfIterations <= 1000 && confidenceValue <= 1.0);
    cout << "computing simrank : \n";
    compute_simrank (Graph, noOfVertices, MaxNoOfIterations, confidenceValue);

    // system("python numpy_test.py"); // generates convergence graph

    return 0;
}