#include "hip/hip_runtime.h"
#include "array_operations.h"
#include "convergeGPU.h"
#include "include_files.h"
#include "cuda_operations_simrank.cuh"
// #include <__clang_cuda_runtime_wrapper.h>
#include <ctime>
#include <system_error>

void ShowMessage() {
    cout << "Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : " << defaultConfidenceValue 
            << "\n\t3. [No. of Iterations] : " << defaultMaxIterations << "\n";
}


// Variables in Unified Memory
__managed__ double ConfidenceValue_;
__managed__ int ThreadCount_, BlockCount_;


__global__
void computeForAPairNodes (int *graph, int verticesCount, double *currentSimRankMtx, double *futureSimRankMtx, int *in_neighbours) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // __printInt(id); // checking current thread.
    if (id > verticesCount * verticesCount) return;
    // int gridStride = gridDim.x * blockDim.x;
    // for (int i = id; i < verticesCount * verticesCount; i += gridStride) { 
    
        int from, to; // store nodes;
        from = id % verticesCount;
        to = id / verticesCount;
        
        // BASE CONDITIONS --> thread break//
        if (from == to) {
            futureSimRankMtx[from * verticesCount + to] = 1.0;
            return;
        }
        
        // printf ("pair formed : (%d , %d)\n",from, to);
        
        int count_inNeighbours_FROM;
        count_inNeighbours_FROM = in_neighbours[from * (verticesCount+1) + verticesCount];
        int count_inNeighbours_TO;
        count_inNeighbours_TO = in_neighbours[to * (verticesCount+1) + verticesCount];
        
        if (count_inNeighbours_FROM == 0 || count_inNeighbours_TO == 0) {
            futureSimRankMtx[from * verticesCount + to] = 0.0;
            return;
        }

        long mul_Cnt = (count_inNeighbours_TO * count_inNeighbours_FROM);
        double NORMALISATION_FACTOR = ConfidenceValue_ / (double)(mul_Cnt);

        /********************DEBUG********************/
        // if (from == 2 && to == 4) {
        //     printf ("inNeighbours count; \n %d : %d \n %d : %d\n", from, count_inNeighbours_FROM, to, count_inNeighbours_TO);
        // }
        /********************DEBUG********************/

        // CPU [experimental] -> [faster than GPU] [result.]
        double simrank_computed_in_neighbour = NORMALISATION_FACTOR * computeFromInNeighbours (currentSimRankMtx, 
                                                                        verticesCount, in_neighbours, 
                                                                        from, to, count_inNeighbours_FROM, count_inNeighbours_TO);
        
        // uncomment for verbose output.
        // printf ("future simrank for pair (%d, %d) : %lf\n", from, to, simrank_computed_in_neighbour * NORMALISATION_FACTOR);

        futureSimRankMtx[from * verticesCount + to] = simrank_computed_in_neighbour;

    // }
}

void calculateSimRankForEachPair (double *simrank, int n_vertices, int *graph, double confidenceValue, int iterationCount) {
    // for each pair of nodes
    double *tmpSimRank;
    hipMallocManaged(&tmpSimRank, sizeof(double) * n_vertices * n_vertices);
    
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&noOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, deviceId);
    
    // THREAD CONFIG. GENERATION. --> bad.
    // int sqVertices = n_vertices * n_vertices;
    // if (n_vertices > 1024)
    //     ThreadCount_ = 1024;
    // else ThreadCount_ = n_vertices;
    // BlockCount_ = ceil (sqVertices / ThreadCount_);
    // if (BlockCount_ == 0) BlockCount_ = 1;
    BlockCount_ = n_vertices * n_vertices;
    ThreadCount_ = 1;


    // Pre-compute the in-neighbour matrix
    int *InNeighbours;
    int InNeighboursSize= sizeof(int) * (n_vertices) * (n_vertices + 1);
    hipMallocManaged (&InNeighbours, InNeighboursSize);
    calculateAllInNeighbours(graph,n_vertices,InNeighbours); // calculates and stores all the in-neighbours 

    
    // if (iterationCount == 1){
    //     printf ("in-neighbours calculation : \n");
    //     for (int i = 0; i < n_vertices; i++) {
    //         printf ("%d : ", i);
    //         for (int j = 0; j < n_vertices + 1; j++) {
    //             printf ("%d ", InNeighbours[i * (n_vertices + 1) + j]);
    //             // __printInt(InNeighbours[i * (n_vertices + 1) + j]);
    //         }
    //         __nl;
    //     }
    // }
    /********************DEBUG********************/
    // seeGraph<int>(InNeighbours,n_vertices); // wrong.
    // int debugNode = 4;
    // printf ("in-neighbours for node - %d\n", debugNode);
    // for (int i = 0; i < (n_vertices+1); i++) {
    //     printf ("%d ", InNeighbours[debugNode * (n_vertices+1) + i]);
    // }printf ("\n");

    // printf ("ThreadCount : %d & BlockCount : %d\n", ThreadCount_, BlockCount_);
    // printf ("computeForAPairNodes() call.\n");
    /********************DEBUG********************/

    float start,end;
    // printf ("current iteration : %d\n", iterationCount);
    start = __time;
    computeForAPairNodes <<< BlockCount_, ThreadCount_ >>> (graph, n_vertices, simrank, tmpSimRank, InNeighbours);
    end = __time;
    totalTime += (float)(end - start) / CLOCKS_PER_SEC;
    hipDeviceSynchronize();

    /********************DEBUG********************/
    // printf ("iteration : #%d \n", iterationCount);
    // seeSimrank(tmpSimRank, n_vertices);
    // printf ("\n\n");
    /********************DEBUG********************/
    
    // copyArr<double>(tmpSimRank, simrank, n_vertices); // iteration complete. 

    // COPY ARRAY TO ORIGINAL.
    for (int i = 0; i < n_vertices; i++) {
        for (int j = 0; j < n_vertices; j++) {
            simrank[i * n_vertices + j] = tmpSimRank[i * n_vertices + j];
        }
    }


    return; 
}

void compute_simrank (int *graph, int noOfVertices, int noOfIterations, double confidenceValue) {
    double *simrank;
    hipMallocManaged(&simrank, sizeof(double) * noOfVertices * noOfVertices);
    initGraph<double> (simrank, noOfVertices, 0.0);

    for(int i = 0; i < noOfVertices; i++) {
        for(int j = 0; j < noOfVertices; j++) {
            simrank[i*noOfVertices+j] = 0.0 + 1.0 * (i==j);
        }
    }

    double normValue=0.0; // for convergence calculation // donot change.
    int currentIteration = 1;

    ConfidenceValue_ = confidenceValue;
    checkConvergence(simrank, noOfVertices, &normValue, "L1");
    printf("starting norm value: %lf\n", normValue);
    while (currentIteration <= noOfIterations) {
        storeNorm(simrank, noOfVertices, "L1");
        storeNorm(simrank, noOfVertices, "L2");

        calculateSimRankForEachPair(simrank, noOfVertices, graph, confidenceValue, currentIteration); // for each (i, j) from |V x V|


        // after experimentation; min value of currerntIteration = 3 can be used.
        if (currentIteration > 3 && checkConvergence(simrank, noOfVertices, &normValue, "L1")) {
            // using L1 Norm for convergence
            break;
        }
        // printf ("calculateSimRankForEachPair() call.\n");
        // printf ("iteration %d\n", currentIteration);
        // seeSimrank(simrank, noOfVertices);
        // printf ("\n\n");
        ++currentIteration;
    }


    /************VERBOSE OUTPUT************/
    seeSimrank(simrank, noOfVertices); // print ans.
    // cout << "converged!\n";
    printf ("converged! @%d\n", currentIteration);
    /************VERBOSE OUTPUT************/
}


int *TakeGraphInput(int *vertices, int *edges, string fileName) {
    ifstream filePtr(DATASET_FOLDER + fileName);
    filePtr >> *vertices;
    filePtr >> *edges;

    int from, to, idx=0, N = *vertices;

    cout << "\nEntered Graph Configuration : \n\tnoOfVertices : " << *vertices << "\n\tnoOfEdges : " << *edges << "\n";

    // storing graph in unified memory.
    int *graph;
    hipMallocManaged(&graph, N * N);
    initGraph<int>(graph, N, 0); // initialise graph to zero
 
    while (idx < *edges) {
        filePtr >> from;
        filePtr >> to;
        graph[from * N + to] = 1;
        ++idx;
    }
    return graph;
}

int main() {

    // Graph Generation.
    // system ("bash start_simrank.sh gnm_random_graph.py");

    __fileIO(); // file input output.

    ShowMessage();
    // system("./delete_l1_l2.sh");

    int noOfVertices, noOfEdges;
    int *Graph;
    Graph = TakeGraphInput(&noOfVertices, &noOfEdges, "graph_input.txt");
    // seeGraph<int>(Graph, noOfVertices);

    int MaxNoOfIterations;
    double confidenceValue;
    TakeSimRankConfigurationInput(MaxNoOfIterations, confidenceValue);
    assert (MaxNoOfIterations <= 1000 && confidenceValue <= 1.0 && (confidenceValue < 0 && confidenceValue != -1));
    cout << "computing simrank : \n";
    compute_simrank (Graph, noOfVertices, MaxNoOfIterations, confidenceValue);

    // system("python numpy_test.py"); // generates convergence graph

    cout << "time taken : ";
    __printFloat(totalTime);

    return 0;
}