
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdio.h>
// #include <math.h>
#include <fstream>
using namespace std;
#define ll long long
#define ull unsigned long long
__global__ 
void kernel (ull *a, int n) {
    int fid = threadIdx.x + blockDim.x * blockIdx.x, tid = threadIdx.x;
    // printf ("current thread[%d]\n", tid);
    for (ull s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            a[fid] += a[fid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        a[blockIdx.x] = a[fid];
    }

}

ull cpukernel (ull *a, int n) {
    ull sum = 0;
    for (ull i = 0; i < n; i++)
        sum += a[i];
    return sum;
}


int main() {
   #ifndef ONLINE_JUDGE
    freopen("test.txt", "r", stdin); 
   #endif

    ull n;
    scanf ("%llu", &n);
    ull *a;
    hipMallocManaged (&a, sizeof(ull) * n);
    // int size = sizeof(a) / sizeof(int);
    for (ull i = 0; i < n; ++i) {
        a[i] = i + 1;
    }
    // printf("original : ");
    // for (int i = 0; i < n; i++) printf("%d ", a[i]);
    
    // 32 blocks of 32 threads.

    float start,end;
    // cpu call
    start = clock();
    ull result = cpukernel(a, n);
    end = clock();
    
    float timeCPU = (float) (end - start) / CLOCKS_PER_SEC;

    start = clock();
    kernel <<< 1024, 1024 >>> (a, n);
    kernel <<< 1, 1024 >>> (a, n);
    end = clock();
    hipDeviceSynchronize();

    float timeGPU = (float) (end - start) / CLOCKS_PER_SEC;    

    // printf("after reduction : ");
    // for (int i = 0; i < n; i++) printf("%d ", a[i]);


    ofstream filePtr("./CPU_Time.txt", std::ios::app);
    ofstream filePtr_ ("./GPU_Time.txt", std::ios::app);

    printf("\nTime elapsed[CPU] : %lf\n", timeCPU);
    printf("\nTime elapsed[GPU] : %lf\n", timeGPU);
    printf("%llu\n", a[0]);

    filePtr << timeCPU << " ";
    filePtr_ << timeGPU << " ";

    filePtr.close();
    filePtr_.close();
    return 0;
}