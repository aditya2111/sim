
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 100
#define SIZE_INT sizeof(int)
#define SIZE (SIZE_INT * N)


__global__
void kernel (int *a, int *b, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    for (int i = id; i < size; i+=gridStride) {
        b[i] = a[i];
    }
    // __syncthreads();
}


int main() {
    #ifndef ONLINE_JUDGE
        freopen("input.txt", "r", stdin);
    #endif
    int *a;
    int *b;
    hipMallocManaged(&a, SIZE);
    hipMallocManaged(&b, SIZE);

    for (int i = 0; i < N; i++) {
        a[i] = N - i + 1;
    }

    int device;
    hipGetDevice(&device);

    int noOfSMs;
    hipDeviceGetAttribute(&noOfSMs, hipDeviceAttributeMultiprocessorCount, device);

    printf("no of sm : %d\n", noOfSMs);

    kernel <<< 32 * noOfSMs, 1 >>> (a, b, N);
    hipDeviceSynchronize();

    printf("a : ");
    for (int i = 0; i < N; i++)
        printf("%d ", a[i]);
    printf("\n\nb : ");
    for (int i = 0; i < N; i++) {
        printf("%d ", b[i]);
    }

    return 0;
}