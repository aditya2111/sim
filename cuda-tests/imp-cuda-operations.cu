#ifndef CUDA_OPS_SIMRANK_H
#define CUDA_OPS_SIMRANK_H
#include <stdio.h>
#define input(a) scanf("%d", &a)
#define output_(a) printf("%d ", a);
#define nl printf("\n")
void see(int *graph, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            output_(graph[i * n + j]);
        }nl;
    }
}

int main() {
    #ifndef ONLINE_JUDGE
        freopen("input.txt", "r", stdin);
    #endif
    int n, e;
    scanf ("%d%d", &n, &e);
    int *graph;
    hipMallocManaged(&graph, sizeof(int) * n * n);
    for (int i = 0; i < e; i++) {
        int from, to;
        input(from);
        input(to);
        graph[from * n + to] = 1;
    }
    

    see(graph, n);

}

#endif