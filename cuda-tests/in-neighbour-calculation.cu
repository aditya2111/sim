
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdio.h>
#include <fstream>

int* readGraph(int *v, int *e) {
    std::ifstream file ("./input.txt");
    file >> *v;
    file >> *e;
    
    int *graph, vert = *v;
    hipMallocManaged(&graph, sizeof(int) * (vert * vert));
    int cnt = *e;
    while (cnt--) {
        int from, to;
        file >> from;
        file >> to;
        graph[from * vert + to] = 1;
    }
    return graph;
}

void see(int *graph, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", graph[i * n + j]);
        }printf("\n");
    }
}

__device__ int countIn_A, countIn_B;

__global__ 
void calculateInNeighbours (int *graph, int size, int *in_neighbours, int node, int* count) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = id; i < size; i += (gridDim.x * blockDim.x)) {
        in_neighbours[i] = graph[i * size + node];
        countIn_A += (in_neighbours[i] == 1);
    }
    // __syncthreads();

}

void CPUInNeighbour (int *graph, int size, int node, int *in_neighbours) {
    for (int i = 0; i < size; i++) {
        in_neighbours[i] = graph[i * size + node];
    }
}

int main() {
    int vert,edges;
    int *graph = readGraph(&vert, &edges);
    // printf("graph : \n");
    // see(graph, vert);
    printf("nodes ranges from 0 to %d\n", vert);
    // int node;
    // printf("calculating in-neighbours for node : ");
    // scanf("%d", &node);

    int node;
    std::ifstream fileptr("./test.txt");
    fileptr >> node;
    printf("node choosen : %d\n", node);
    int *in_neighbours;
    hipMallocManaged(&in_neighbours, sizeof(int) * vert);

    int deviceId;
    hipGetDevice(&deviceId);
    int noOfSms;
    hipDeviceGetAttribute(&noOfSms, hipDeviceAttributeMultiprocessorCount, deviceId);
    int *count;
    hipMallocManaged(&count, sizeof(int));

    float start, end;
    start = clock();
    calculateInNeighbours <<< 32 * noOfSms, 1023 >>> (graph, vert, in_neighbours, node, count);
    end = clock();
    hipDeviceSynchronize();
    // end = clock();

    int *in_neigh2;
    hipMallocManaged(&in_neigh2, sizeof(int) *vert);

    float start2, end2;
    start2 = clock();
    CPUInNeighbour(graph, vert, node, in_neigh2);
    end2 = clock();

    float time_ = (float)(end2 - start2) / CLOCKS_PER_SEC;
    float gputime_ = (float)(end - start) / CLOCKS_PER_SEC;

    // printf("in-Neighbours [GPU] : ");
    // for (int i = 0; i <vert; i++) {
    //     printf("%d ", in_neighbours[i]);
    // }printf("\n");

    // printf("in-Neighbours [CPU] : ");
    // for (int i = 0; i <vert; i++) {
    //     printf("%d ", in_neigh2[i]);
    // }printf("\n");

    std::ofstream cputime ("./CPU_Time.txt", std::ios::app);
    std::ofstream gputime ("./GPU_Time.txt", std::ios::app);



    // printf("time [GPU] : %lf\n",(float)(end - start) / CLOCKS_PER_SEC);
    // printf( "time [CPU] : %lf\n", time_);

    cputime << time_ << " ";
    gputime << gputime_ << " ";



    return 0;
}