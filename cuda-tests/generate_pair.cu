
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void kernel (int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = blockDim.x * gridDim.x;
    for (int i = id; i < n * n; i+=gridStride) {
        int from, to; 
        from = id % n;
        to = id / n;
        printf ("pair : (%d, %d)\n", from, to);
    }
    
}

int main() {
    freopen64 ("output.txt", "w", stdout);
    int n = 5; // pair of (0 to 5)
    int Block_ = n * n;
    int Thread_ = 1;

    kernel <<< Block_, Thread_ >>> (n);
    hipDeviceSynchronize();

    return 0;
}