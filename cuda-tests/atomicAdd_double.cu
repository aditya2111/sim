
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__global__ void kernel (double* arr, double* sum) {
    int id = threadIdx.x;
    atomicAdd(&sum[0], arr[id]);
}

int main() {
    double arr[N];
    double* d_arr;
    for(int i = 0; i < N; i++)
        arr[i] = 11.0;

    double sum[1]; sum[0] = 0.0;
    double* d_sum;

    hipMalloc(&d_sum, sizeof(double));
    hipMemcpy(d_sum, sum, sizeof(double), hipMemcpyHostToDevice);

    hipMalloc(&d_arr, sizeof(double) * N);
    hipMemcpy(d_arr, arr, sizeof(double) * N, hipMemcpyHostToDevice);

    kernel<<<1, N>>>(d_arr, d_sum);
    
    hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(arr, d_arr, sizeof(double) * N, hipMemcpyDeviceToHost);
    
    printf("arr : \n");
    for(int i=0;i<N;i++) {
        printf("%lf ", arr[i]);
    }
    printf("\nsummation : %lf\n", sum[0]);

    return 0; 
}
