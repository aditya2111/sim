
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__
void kernel () {
    printf("Thread[%d] Run.\n", threadIdx.x);
}


void callKernel (int n) {
    kernel <<< 1, n >>> ();
    hipDeviceSynchronize();
}
