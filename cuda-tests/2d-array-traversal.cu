
#include <hip/hip_runtime.h>
#include <stdio.h>

__managed__ int ThreadCount_, BlockCount_;


__global__ 
void kernel (int *a, int size) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < size * size) {
        int from, to;
        from = id % size;
        to = id / size;
        printf("pair : (%d, %d) -> %d\n", from, to, a[from * size + to]);
    }
}

int main() {
    int size = 3;
    int *a; // 3 x 3 matrix;
    hipMallocManaged(&a, sizeof(int) * size * size);

    for (int i = 0; i < size * size; i++) a[i] = i + 1;

    ThreadCount_ = 10;
    BlockCount_ = 1;

    kernel <<< BlockCount_, ThreadCount_ >>> (a, size);
    hipDeviceSynchronize();
    return 0;
}