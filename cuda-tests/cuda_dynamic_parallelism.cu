#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 50
#define SIZE sizeof(int) * N
#define print(a) printf("%d ", a)
#define nl printf("\n")

__global__ void kernel3 (int *a) {
    int id = threadIdx.x;
    a[id] = id * 10;

    printf("kernel3()! \t a[%d] : %d\n", id, a[id]);

}

__global__ void kernel2 (int *a) {
    int id = threadIdx.x;
    a[id] = id + 1;

    printf("kernel2()! \t a[%d] : %d\n", id, a[id]);
}

__global__ void kernel1 (int *a, int M) {
    __syncthreads(); // block level synchronization
    int id = threadIdx.x;
    if (id == 0) {
        // printf ("current thread : %d\n", id);
        kernel2 <<< 1, M >>> (a);
        hipDeviceSynchronize();
        // printf("\n===================\n");
    }
    __syncthreads();
    if (id == 1) {
        // printf("current thread : %d\n", id);
        kernel3 <<< 1, M >>> (a);
        hipDeviceSynchronize();
    }
    __syncthreads();
}

int main() {
    int *a;
    hipMallocManaged(&a, SIZE); // stored in unified.
    // printf("original : ");
    // for (int i = 0; i < N; i++) {
    //     print(a[i]);
    // }nl;
    kernel1 <<< 1 , N >>> (a, N);
    hipDeviceSynchronize();
    // printf("after kernel call! : ");
    // for(int i = 0; i < N; i++) {
    //     print(a[i]);
    // }nl;
    return 0;
}