
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 5

__global__ void kernel(int *arr, int *pos1, int *pos2, int* summation) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&summation[0], arr[pos1[id] * N + pos2[id]]);
}

int main(){
    int a[N*N];
    for(int i = 0; i < N;i++) {
        for(int j = 0; j < N; j++) {
            a[i*N+j] = rand() % N;
        }
    }

    printf("array generated : \n");
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            printf("%d ", a[i*N+j]);
        }printf("\n");
    }

    int pos1[4], pos2[4];
    for(int i = 0; i < 4; i++) {
        pos1[i] = rand() % N;
        pos2[i] = rand() % N;
    }

    printf(" new points : \n");
    for(int i = 0; i < 4; i++) {
        printf("%d\t%d\n", pos1[i], pos2[i]);
    }
    
    int* d_a, *d_pos1, *d_pos2;
    hipMalloc(&d_a, sizeof(int) * N * N);
    hipMalloc(&d_pos1, sizeof(int) * 4);
    hipMalloc(&d_pos2, sizeof(int) * 4);

    hipMemcpy(d_a, a, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_pos1, pos1, sizeof(int) * 4, hipMemcpyHostToDevice);
    hipMemcpy(d_pos2, pos2, sizeof(int) * 4, hipMemcpyHostToDevice);
    
    int sum[1]; sum[0]=0;
    int* d_sum;
    hipMalloc(&d_sum, sizeof(int));
    hipMemcpy(d_sum, sum, sizeof(int), hipMemcpyHostToDevice);

    kernel<<<4,4>>>(d_a, d_pos1, d_pos2, d_sum);

    hipMemcpy(sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nsummation : %d\n", sum[0]);
    
    return 0;
}
