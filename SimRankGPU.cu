#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <unordered_map>
#include "convergeGPU.h"
#include <fstream>
#include <stdio.h>
#include "array_operations.h"
#include <hip/hip_runtime_api.h>
#include <thrust/scan.h>
#include <thrust/count.h>

#define matrix_INT vector<vector<int>>
#define matrix_DOUBLE vector<vector<double>>
#define ROW_INT vector<int>
#define ROW_DOUBLE vector<double>
#define newline printf("\n")

/* File Output
ofstream fout;
fout.open("output_simrank.txt");
****************/

void Message() {
    printf("Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : 0.9\n\t3. [No. of Iterations] : 1000\n");
}

/* GPU Kernel */
__global__ void calculateSimRankPair (int *A, int *B, double *simrank, double *summation, int* n_Vertices) {
    atomicAdd(&summation[0], simrank[A[blockIdx.x] * n_Vertices[0] + B[threadIdx.x]]);
}
/*************/

__device__ int n_vertices_gpu;
__device__ int inNeighbourIndex; // stores the in-neighbour index for further calculation.
__device__ int *calculationArray;

__device__ int inAsize, inBsize, tmpCalculation; // global; device declaration.
__device__ double tmpSimrankComputation; // stores temporary simrank values.

__global__ void computeSimrank (int* Graph, double* SimRank, int* inNeighbours, int node_from, int node_to) {
    int bid = blockIdx.x;
    //int ia_neighbour = blockIdx.x, 
    //    ib_neighbour = blockIdx.y;

    int from = node_from,
        to = node_to;
    int ia_neighbour = bid / n_vertices_gpu, 
        ib_neighbour = bid % n_vertices_gpu;
    
   
    if (inNeighbours[from * (n_vertices_gpu + 1) + ia_neighbour] == 1 && inNeighbours[to * (n_vertices_gpu + 1) + ib_neighbour] == 1) {
       // printf("inNeighbours : %d and %d\n", ia_neighbour, ib_neighbour);
        tmpSimrankComputation = SimRank[ia_neighbour * n_vertices_gpu + ib_neighbour];  
    }

}


__global__ void kernel_ (int *Graph, int *noOfVertices, double *confidence_value, double *simrankCurrent, double *simrank, int *inNeighbours) {
   n_vertices_gpu = noOfVertices[0]; // global device variable : # of vertices
   int bid = blockIdx.x;
   int node_from, node_to;

   node_from = bid / n_vertices_gpu;
   node_to = bid % n_vertices_gpu;
    
   //printf("%d and %d\n", node_from, node_to);

   /*if (node_from == node_to) {
      simrankCurrent[node_from * n_vertices_gpu + node_to] = 1.0; 
    return;
   }*/

   simrankCurrent[node_from * n_vertices_gpu + node_to] = (node_from == node_to) * 1.0 + (node_from != node_to) * 0.0;


   if (node_from != node_to) {
       tmpSimrankComputation = 0.0; // stores the temporary simrank computaion for any 2 nodes. 
   
        computeSimrank <<< n_vertices_gpu * n_vertices_gpu, 1 >>> (Graph, simrank, inNeighbours, node_from, node_to);
        hipDeviceSynchronize();
   
        double mul = inNeighbours[node_from * (n_vertices_gpu + 1) + n_vertices_gpu] * inNeighbours[node_to * (n_vertices_gpu + 1) + n_vertices_gpu];
        //simrankCurrent[node_from * n_vertices_gpu + node_to] = (mul > 0) * tmpSimrankComputation * (confidence_value[0] / mul) + (mul == 0) * 0;
        if (mul == 0) {
            simrankCurrent[node_from * n_vertices_gpu + node_to] = 0.0;
        }
        else {
            simrankCurrent[node_from * n_vertices_gpu + node_to] = tmpSimrankComputation * (confidence_value[0] / mul);
        }
    }
}


double totalKernelTime = 0.0;
// updated kernel call.
void SimRankForAllNodes_ (int iteration, double *SimRank, int *Graph, int n_vertices, double confidence_value) {
    double *tmpSimrank = createArray_D(n_vertices);
    
    clock_t startKernel, endKernel; // timing the external kernel.

    int n_blocks = n_vertices;
    
    int sizeofSimRank = n_vertices * n_vertices;    

    /* Device Parameters */

    double *device_tmpSimrank;
    hipMalloc(&device_tmpSimrank, sizeof(double) * sizeofSimRank);
    hipMemcpy(device_tmpSimrank, tmpSimrank, sizeof(double) * sizeofSimRank, hipMemcpyHostToDevice);

    double *device_currentSimrank;
    hipMalloc(&device_currentSimrank, sizeof(double) * sizeofSimRank);
    hipMemcpy(device_currentSimrank, SimRank, sizeof(double) * sizeofSimRank, hipMemcpyHostToDevice);
        
    int vertex[1] = {n_vertices};
    int *device_vertex;
    hipMalloc(&device_vertex, sizeof(int));
    hipMemcpy(device_vertex, vertex, sizeof(int), hipMemcpyHostToDevice);

    int *device_Graph;
    hipMalloc(&device_Graph, sizeof(int) * sizeofSimRank);
    hipMemcpy(device_Graph, Graph, sizeof(int) * sizeofSimRank, hipMemcpyHostToDevice);

    double cv[1] = {confidence_value};
    double *device_cv;
    hipMalloc(&device_cv, sizeof(double));
    hipMemcpy(device_cv, cv, sizeof(double), hipMemcpyHostToDevice);
    
    /* Pre-Computing the in-neighbours */
    int *in_neighbours;
    in_neighbours = findInNeighbours_(Graph, n_vertices); // <array_operations.h>
    // print the in-neighbours
    //printInNeighbours (in_neighbours, n_vertices); <array_operations.h>
   
    int* device_in_neighbours;
    hipMalloc(&device_in_neighbours, sizeof(int) * n_vertices * (n_vertices + 1));
    hipMemcpy(device_in_neighbours, in_neighbours, sizeof(int) * n_vertices * (n_vertices + 1), hipMemcpyHostToDevice);

    /* Kernel Call */ 
    startKernel = clock();
    kernel_ <<< n_blocks * n_blocks, 1 >>>(device_Graph, device_vertex, device_cv, device_tmpSimrank, device_currentSimrank, device_in_neighbours);
    endKernel = clock();
    
    totalKernelTime += (endKernel - startKernel) / CLOCKS_PER_SEC;

    hipMemcpy(tmpSimrank, device_tmpSimrank, sizeof(double) * sizeofSimRank, hipMemcpyDeviceToHost); 
    
    for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            SimRank[i * n_vertices + j] = tmpSimrank[i * n_vertices + j];
        }    
    }


    /*for (int i = 0; i < n_vertices; i++) {
        for (int j = 0; j < n_vertices; j++) {
            printf("%d ", SimRank[i * n_vertices + j]);
        }
        printf("\n");
    }*/
}

void SimRankForAllNodes(int iteration, double* SimRank, int** Graph, int n_vertices, double confidence_value) {
   double* tmpSimrank = createArray_D(n_vertices); // default initialisation with 0.0 as array values. 
   
   /* Kernel Timing */
   clock_t startKernel, endKernel;
   //double totalKernelTime = 0.0;

   for(int i = 0; i < n_vertices; i++) {
       //hipStream_t streams[n_vertices];
       for(int j = 0; j < n_vertices; j++) {
           // GPU Computation.
           /* base condition: [1] Same Node */
           if(i == j) { 
               tmpSimrank[i * n_vertices + j] = 1.0;
               //printf("base case - 1[same node]\n");
               continue;
           }
           /* this code has some problem */
           int ia_size, ib_size;
           int* I_A = findInNeighbors(i, Graph, n_vertices, &ia_size);
           int* I_B = findInNeighbors(j, Graph, n_vertices, &ib_size);
           /******************************/
           /* Normalisation Factor */ 
           double normalisation_factor = confidence_value / (ia_size * ib_size);
            
           /* Base Condition : [2] No In_Neighbours of one of the edges */
           if(ia_size == 0 || ib_size == 0) {
               tmpSimrank[i * n_vertices + j] = 0.0;
               //printf("base case - 2[no in-neighbors]\n");
               continue;
           }

           /* GPU PARAMETERS */
           int n_CUDA_threads = ib_size;
           int n_CUDA_blocks = ia_size;

           //printf("GPU PARAMETERS :\t # of blocks : %d\t # of threads : %d\n", n_CUDA_blocks, n_CUDA_threads);
           /******************/ 

           /* not used.
              int nodes[2];
           nodes[0] = i;
           nodes[1] = j;

           int* device_nodes;
           hipMalloc(&device_nodes, sizeof(int) * 2); // 2 integers are stored, the #from node and the #to node
           hipMemcpy(device_nodes, nodes, sizeof(int) * 2, hipMemcpyHostToDevice);
           */
           int* d_I_A, *d_I_B;
           
           /* nv profiling */
           //hipProfilerStart();
           
           hipMalloc(&d_I_A, sizeof(int) * ia_size);
           hipMalloc(&d_I_B, sizeof(int) * ib_size);
           hipMemcpy(d_I_A, I_A, sizeof(int) * ia_size, hipMemcpyHostToDevice);
           hipMemcpy(d_I_B, I_B, sizeof(int) * ib_size, hipMemcpyHostToDevice);
            
           int n_simrank = n_vertices * n_vertices;
           double* d_simrank;
           hipMalloc(&d_simrank, sizeof(double) * n_simrank);
           hipMemcpy(d_simrank, SimRank, sizeof(double) * n_simrank, hipMemcpyHostToDevice);

           double tmp_ans[1]; tmp_ans[0] = 0.0;
           double* d_ans;
           hipMalloc(&d_ans, sizeof(double));        
           hipMemcpy(d_ans, tmp_ans, sizeof(double), hipMemcpyHostToDevice);

           int d_vert[1]; d_vert[0] = n_vertices;
           int *device_n_vertices;
           hipMalloc(&device_n_vertices, sizeof(int));
           hipMemcpy(device_n_vertices, d_vert, sizeof(int), hipMemcpyHostToDevice); 
 
           /* kernel call */
           startKernel = clock();
           calculateSimRankPair<<<n_CUDA_blocks, n_CUDA_threads>>>(d_I_A, d_I_B, d_simrank, d_ans, device_n_vertices);
           endKernel = clock();
           /***************/

           totalKernelTime += (double)(endKernel - startKernel) / CLOCKS_PER_SEC;
           //hipDeviceSynchronize(); 
           hipMemcpy(tmp_ans, d_ans, sizeof(double), hipMemcpyDeviceToHost);  

           // nv profiler end
           //hipProfilerStop();
           
           tmpSimrank[i * n_vertices + j] = tmp_ans[0] * normalisation_factor;
       }
   } 
   
    for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            SimRank[i * n_vertices + j] = tmpSimrank[i * n_vertices + j];
        }
    }

    // debug each iteration.
    //printf("Simrank updated!\n");
    /*for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            printf("%lf ", SimRank[i * n_vertices + j]);
        }
        printf("\n");
    }*/
}
void ComputeSimRankMatrix (int* Graph, int noOfVertices, int noOfEdges, int max_iterations, double confidence_value) {
    double* SimRank = (double*) malloc(sizeof(double) * noOfVertices * noOfVertices);
    int V = noOfVertices;
    // 1 st iterations.
    for(int i = 0; i < V; i++) {
        for(int j = 0; j < V; j++) {
            SimRank[i*V+j] = 0.0 + 1.0 * (i==j);
        }
    } 
    double normValue = 0.00;
    checkConvergence(SimRank, V, &normValue);
    // rest of the iterations/
    int k = 1;
    for(; k<max_iterations; k++) {
    //printf("iteration : #%d\n", k);
        /* below two functions are for plotting convergence graph */
        storeL2Norm(SimRank, noOfVertices);
        storel1Norm(SimRank, noOfVertices);

        SimRankForAllNodes_(k, SimRank, Graph, noOfVertices, confidence_value);
        /* Checking Convergence of SimRank Matrix */ 
        if (k > 2 && checkConvergence(SimRank, noOfVertices, &normValue) == true) {
            break;
        }
    }
    //printf("Total Kernel Time : %.5f\n",totalKernelTime); 
    printf("Converged on : %d\n",k);
    
    printf("SimRank Algorithm Converged!\nFinal SimRank Matrix : \n");
    for(int i = 0; i < noOfVertices; i++) {
        for(int j = 0; j < noOfVertices; j++) {
            printf("%.4f ", SimRank[i*noOfVertices+j]);
        }printf("\n");
    }
    printf("\n");
}

int* TakeInput(int *V, int *E) {
    string filePath = "./tests/datasets/";
    //string fileName = "watts_strogatz.txt";
    string fileName = "graph_input.txt";
    ifstream file(filePath + fileName);
    
    //ifstream file("input.txt");
    file >> *V;
    file >> *E;
    
    int n_vertices = *V, n_edges = *E;
    int from, to;
    int idx = 0;
    
    printf("\nEntered Graph Configuration : \n");
    printf("\tnoOfVertices: %d\n\tnoOfEdges: %d\n",*V,*E);  
    
    int *Graph; 
    Graph = (int*) malloc(sizeof(int) * n_vertices * n_vertices);
    
    /* int** Graph = new int*[n_vertices+1];
    for(int i = 0; i < n_vertices; i++) {
        Graph[i] = new int[n_vertices+1];
    }*/
    
    for (int i = 0; i < n_vertices; i++) {
        for (int j = 0; j < n_vertices; j++) {
           Graph[i * n_vertices + j] = 0; 
        }
    }
    

    while(idx < n_edges) {
        file >> from;
        file >> to;
        Graph[from * n_vertices + to] = 1;
        idx++; 
    }
    return Graph;
}

void TakeSimRankConfigurationInput(int &iterations, double &confidence) {
    printf("Enter no. of iterations[for default, input -1]: ");
    scanf("%d",&iterations);
    printf("Enter Confidence-Value[0-1, for default, input -1]: ");
    scanf("%lf",&confidence);

    if(iterations == -1) iterations = 1000;
    if(confidence == -1) confidence = 0.9;

    cout << "\n*SimRank Configuration Chosen: \n\tIterations: " << iterations << "\n\tConfidence Value: " << confidence << "\n";
}

int main() {
    Message();
    
    //Deleting the l1 norm parameters, for future creation.unavoidable 
    system("./delete_l1_l2.sh"); 
    
    // Graph Input.
    int noOfVertices, noOfEdges;
    // converting to 1-d array
    //int** Graph = TakeInput(&noOfVertices, &noOfEdges);
    int *Graph = TakeInput (&noOfVertices, &noOfEdges); 

    // see graph
    for (int i = 0; i < noOfVertices; i++) {
        for (int j = 0; j < noOfVertices; j++) {
            printf("%d ", Graph[i * noOfVertices + j]);
        }newline;
    }

    // Input of SimRank Configuration parameters
    int noOfIterations;
    double confidence_value;
    TakeSimRankConfigurationInput(noOfIterations, confidence_value);
    
    // Time Calculation for Whole Computation.
    //clock_t startTime,endTime;
    
    //startTime = clock();
    ComputeSimRankMatrix(Graph, noOfVertices, noOfEdges, noOfIterations, confidence_value);
    //endTime = clock();
    
    //float time2 = (float)(endTime - startTime) / CLOCKS_PER_SEC;
    printf("[GPU]Time Elapsed in seconds: %.4f\n", totalKernelTime);
    
    //Generating Convergence Graph. 
    //system("python numpy_test.py");
    return 0;
}
