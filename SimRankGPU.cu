#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <unordered_map>
#include "converge.h"
#include <fstream>
#include <stdio.h>
#include "array_operations.h"

#define matrix_INT vector<vector<int>>
#define matrix_DOUBLE vector<vector<double>>
#define ROW_INT vector<int>
#define ROW_DOUBLE vector<double>

/* File Output
ofstream fout;
fout.open("output_simrank.txt");
****************/

void Message() {
    printf("Default Configuration : \n\t1. [Directed-Graph]\n\t2. [Confidence Value] : 0.9\n\t3. [No. of Iterations] : 1000\n");
}
__global__ void calculateSimRankPair (int *A, int *B, double *simrank, double *summation, int* n_Vertices) {
    int from = blockIdx.x;
    int to = threadIdx.x;
    
    atomicAdd(&summation[0], simrank[A[from] * n_Vertices[0] + B[to]]);

}
void SimRankForAllNodes(int iteration, double* SimRank, int** Graph, int n_vertices, double confidence_value) {
   double* tmpSimrank = createArray_D(n_vertices); // default initialisation with 0.0 as array values. 
   
   for(int i = 0; i < n_vertices; i++) {
       for(int j = 0; j < n_vertices; j++) {
           // GPU Computation.
           /* base conditions */
           if(i == j) { 
               tmpSimrank[i * n_vertices + j] = 1.0;
               continue;
           }

           int* I_A = findInNeighbors(i, Graph, n_vertices);
           int* I_B = findInNeighbors(j, Graph, n_vertices);
           int ia_size = sizeof(I_A) / sizeof(int);
           int ib_size = sizeof(I_B) / sizeof(int);

           if(ia_size == 0 || ib_size == 0) {
               tmpSimrank[i * n_vertices + j] = 0.0;
               continue;
           }

           /* GPU PARAMETERS */
           int n_CUDA_threads = ib_size;
           int n_CUDA_blocks = ia_size;
           /******************/ 

           int nodes[2];
           nodes[0] = i;
           nodes[1] = j;

           int* device_nodes;
           hipMalloc(&device_nodes, sizeof(int) * 2); // 2 integers are stored, the #from node and the #to node
           hipMemcpy(device_nodes, nodes, sizeof(int) * 2, hipMemcpyHostToDevice);
           
           int* d_I_A, *d_I_B;
           hipMalloc(&d_I_A, sizeof(int) * ia_size);
           hipMalloc(&d_I_B, sizeof(int) * ib_size);
           hipMemcpy(d_I_A, I_A, sizeof(int) * ia_size, hipMemcpyHostToDevice);
           hipMemcpy(d_I_B, I_B, sizeof(int) * ib_size, hipMemcpyHostToDevice);
            
           int n_simrank = n_vertices * n_vertices;
           double* d_simrank;
           hipMalloc(&d_simrank, sizeof(double) * n_simrank);
           hipMemcpy(d_simrank, SimRank, sizeof(double) * n_simrank, hipMemcpyHostToDevice);

           double tmp_ans[1]; tmp_ans[0] = 0.0;
           double* d_ans;
           hipMalloc(&d_ans, sizeof(double));        
           hipMemcpy(d_ans, tmp_ans, sizeof(double), hipMemcpyHostToDevice);
        
           int d_vert[1]; d_vert[0] = n_vertices;
           int *device_n_vertices;
           hipMalloc(&device_n_vertices, sizeof(int));
           hipMemcpy(device_n_vertices, d_vert, sizeof(int), hipMemcpyHostToDevice); 

           calculateSimRankPair<<<n_CUDA_threads, n_CUDA_blocks>>>(d_I_A, d_I_B, d_simrank, d_ans, device_n_vertices);
           
           hipMemcpy(tmp_ans, d_ans, sizeof(double), hipMemcpyDeviceToHost);          
           tmpSimrank[i * n_vertices + j] = tmp_ans[0];
       }
   } 
   
    for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            SimRank[i * n_vertices + j] = tmpSimrank[i * n_vertices + j];
        }
    }
    printf("Simrank updated!\n");
    for(int i = 0; i < n_vertices; i++) {
        for(int j = 0; j < n_vertices; j++) {
            printf("%lf ", SimRank[i * n_vertices + j]);
        }
        printf("\n");
    }
}

void ComputeSimRankMatrix (int** Graph, int noOfVertices, int noOfEdges, int max_iterations, double confidence_value) {
    double* SimRank = (double*) malloc(sizeof(double) * noOfVertices * noOfVertices);
    int V = noOfVertices;
    // 1 st iterations.
    for(int i = 0; i < V; i++) {
        for(int j = 0; j < V; j++) {
            SimRank[i*V+j] = 0.0 + 1.0 * (i==j);
        }
    } 

    // rest of the iterations/
    for(int k=1; k<max_iterations; k++) {
        // convergence will be checked here.
        /**/
        SimRankForAllNodes(k, SimRank, Graph, noOfVertices, confidence_value);
    }

 
    printf("SimRank Algorithm Converged!\nFinal SimRank Matrix : \n");
    for(int i = 0; i < noOfVertices; i++) {
        for(int j = 0; j < noOfVertices; j++) {
            printf("%.4f ", SimRank[i*noOfVertices+j]);
        }printf("\n");
    }
    printf("\n");

}

// leave this function alone.
// void ComputeSimrankMatrix(matrix_INT Graph,int noOfVertices, int noOfEdges, int max_iterations, double confidence_value) {
//     // Optimising for space.
//     //matrix_DOUBLE SimRankCurrent;

//     /* Optimising for GPU */
//     ROW_DOUBLE SimRank_(noOfVertices * noOfVertices, 0.0);

//     //matrix_DOUBLE initMatrix(noOfVertices, ROW_DOUBLE(noOfVertices, 0.0));
    
//     for(int i = 0; i < noOfVertices ; i++) {
//         SimRank_[i * noOfVertices + i] = 1.0;
//     }
   
//     /*
//     cout << "1D SimRank: \n";
//     for(int i = 0; i < noOfVertices ; i++) {
//         for(int j = 0; j < noOfVertices; j++) {
//             printf("%.4f ", SimRank_[i*noOfVertices+j]);
//         }
//         printf("\n");
//     }
//     */
    
//     int k = 1;
//     for(; k < max_iterations; k++) {
//         // Below line - debugging.
//         //cout << "iteration no. -> " << k << "\n";

//         /*
//             For each iterations, we have to see whether it converges or not.
//             See : converge.h
//             *READ MORE*
//         if(checkConvergence(SimRank, confidence_value) == true) {
//             break;
//         }*/
//         SimrankForAllNodes(k, confidence_value, noOfVertices, Graph, SimRank_);
//     }
    
//     printf("SimRank Algorithm Converged!\nFinal SimRank Matrix : \n");
//     for(int i = 0; i < noOfVertices; i++) {
//         for(int j = 0; j < noOfVertices; j++) {
//             printf("%.4f ", SimRank_[i*noOfVertices+j]);
//         }printf("\n");
//     }
//     printf("\n");
// }   
int** TakeInput(int *V, int *E) {
    ifstream file("input.txt");
    file >> *V;
    file >> *E;
    
    int n_vertices = *V, n_edges = *E;
    int from, to;
    int idx = 0;
    
    printf("\nEntered Graph Configuration : \n");
    printf("\tnoOfVertices: %d\n\tnoOfEdges: %d\n",*V,*E);  
    int** Graph = new int*[n_vertices];
    for(int i = 0; i < n_vertices; i++) {
        Graph[i] = new int[n_vertices];
    }
   // matrix_INT Graph(*V, ROW_INT(*V, 0));
    while(idx < n_edges) {
        file >> from;
        file >> to;
        Graph[from][to] = 1;
        idx++; 
    }
    return Graph;
}

void TakeSimRankConfigurationInput(int &iterations, double &confidence) {

    printf("Enter no. of iterations[for default, input -1]: ");
    scanf("%d",&iterations);
    printf("Enter Confidence-Value[0-1, for default, input -1]: ");
    scanf("%lf",&confidence);

    if(iterations == -1) iterations = 1000;
    if(confidence == -1) confidence = 0.9;

    cout << "\n*SimRank Configuration Chosen: \n\tIterations: " << iterations << "\n\tConfidence Value: " << confidence << "\n";
}

int main() {
    Message();
    
    // Graph Input.
    int noOfVertices, noOfEdges;
    //matrix_INT Graph = TakeInput(&noOfVertices,&noOfEdges);
    int** Graph = TakeInput(&noOfVertices, &noOfEdges);

    // SimRank Configuration.
    int noOfIterations;
    double confidence_value;
    TakeSimRankConfigurationInput(noOfIterations, confidence_value);
    ComputeSimRankMatrix(Graph, noOfVertices, noOfEdges, noOfIterations, confidence_value);
    return 0;
}
